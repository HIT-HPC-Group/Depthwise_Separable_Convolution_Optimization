#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <stdlib.h>
#include <iomanip>
#include <time.h>
#include <random>
#include <vector>
#include <fstream>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

//CSV文件头,保存路径
#ifdef AMD_PLATFORM
#define CSVPATH "DCU_Depthwise_result.csv"
// vector<string> csvHeader={"Input Batch","Input Channel","Height","Filter Size","Stride","DCU Kernel(ms)","HipDNN(ms)"};
#else
#define CSVPATH "V100S_Depthwise_result.csv"
// vector<string> csvHeader={"Input Batch","Input Channel","Height","Filter Size","Stride","V100S Kernel(ms)","Cudnn(ms)"};
#endif

/*
CUDA and CUDNN Error Handling

checkCuda(err)  - to check if an CUDA API call returned some error.
checkKernel()   - to check if the kernel invocation is failed.
checkCudnn(err) - to check if an CUDNN API call returned some error.
*/
#define checkCuda(err) __checkCuda(err, __FILE__, __LINE__)
#define checkKernel() __checkKernel(__FILE__, __LINE__)
#define checkCudnn(err) __checkCudnn(err, __FILE__, __LINE__)

inline void __checkCuda(hipError_t err, const char* file, const int line) {
	if (hipSuccess != err) {
		printf("checkCuda() failed at %s : %i : %s\n", file, line, hipGetErrorString(err));
		exit(-1);
	}
}

inline void __checkKernel(const char* file, const int line) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		printf("checkKernel() failed at %s : %i : %s\n", file, line, hipGetErrorString(err));
		exit(-1);
	}
}

inline void __checkCudnn(hipdnnStatus_t err, const char* file, const int line) {
	if (HIPDNN_STATUS_SUCCESS != err) {
		printf("checkCudnn() failed at %s : %i : %s\n", file, line, hipdnnGetErrorString(err));
		exit(-1);
	}
}

/*
writeCsv():
	write to csv
*/
void writeCsv(int batchnumber, int channel, int height, int filterheight, int stride, float kerneltime, float cudnntime) {
	fstream fs;
	fs.open(CSVPATH, ios::app);

	if (!fs)
	{
		//创建文件
		ofstream fout(CSVPATH, ios::app);
		if (fout)
		{
			// fout << csvHeader[0] << ','
			// << csvHeader[1] << ','
			// << csvHeader[2] << ','
			// << csvHeader[3] << ','
			// << csvHeader[4] << ','
			// << csvHeader[5] << ','
			// << csvHeader[6] << std::endl;
			fout << batchnumber << ','
				<< channel << ','
				<< height << ','
				<< filterheight << ','
				<< stride << ','
				<< kerneltime << ','
				<< cudnntime << endl;
			fout.close();
		}
	}
	else
	{
		fs << batchnumber << ','
			<< channel << ','
			<< height << ','
			<< filterheight << ','
			<< stride << ','
			<< kerneltime << ','
			<< cudnntime << endl;
		fs.close();
	}
}

/*
compareOutput():
	Compare the result calculated by our kernel and that by the cuDNN library.
	Use cuDNN library as a reference.
Input:
	n            - batch number
	c            - channel number
	h            - height
	w            - width
	kernelOutput - output data of our kernel
	cudnnOutput  - output data of the cudnn
	delta        - a small value. Allowed numerical differece between each element
Output:
	-1           - our kernel is wrong
	0            - out kernel is correct
*/
int compareOutput(int n, int c, int h, int w, const float* kernelOutput, const float* cudnnOutput, float delta) {
	int i, j, k, l;

	// Loop over each element, and compare the value.
	// If the difference is small, then accept, or, reject and return.
	for (i = 0; i < n; i++) {
		for (j = 0; j < c; j++) {
			for (k = 0; k < h; k++) {
				for (l = 0; l < w; l++) {
					if (abs(kernelOutput[i * c * h * w + j * h * w + k * w + l] - cudnnOutput[i * c * h * w + j * h * w + k * w + l]) > delta) {
						printf("%f, %f\n", kernelOutput[i * c * h * w + j * h * w + k * w + l], cudnnOutput[i * c * h * w + j * h * w + k * w + l]);
						printf("Wrong! Output Batch Idx: %d, Channel Idx: %d, Row Idx: %d, Col Idx: %d\n", i, j, k, l);
						return -1;
					}
				}
			}
		}
	}
	return 0;
}

/*
* warmup()
* To get GPU initialization ready
*/
__global__ void warmup() {
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	float ia, ib;
	ia = ib = 0.0f;
	ib += ia + tid;
}

/*
Depthwise Convolution Kernel.

Case: filter 3 x 3, input 7 x 7, stride 1, padding 1

The number of channel must be multiple of 32.
Used in the MobileNet V2 and EfficientNet B0, in case of
	1) 7 x 7 x 960 -> 7 x 7 x 960, stride = 1, filter = 3
	2) 7 x 7 x 1152 -> 7 x 7 x 1152, stride = 1, filter = 3
*/
__global__ void Filter3x3_Input7x7_Stride1(const float* input, const float* filter, float* output,
	int inputBatchNumber, int inputChannel, int inputHeight, int inputWidth,
	int filterLayerNumber, int filterHeight, int filterWidth,
	int outputBatchNumber, int outputChannel, int outputHeight, int outputWidth,
	int padding, int stride,
	float alpha, float beta) {

	// every 32 channels is a group.
	__shared__ float filterData[32 * 9];	// filter is 3 x 3 = 9
	__shared__ float inputData[32 * 7 * 9]; // original input is 7 x 7, padded to be 9 x 9. ignore up and bottom padding, so 7 x 9

	float inTemp0, inTemp1, inTemp2;
	float sum0, sum1, sum2;  // to accumulate the row sum result. rolling recycle.

	int channelGroupSize = 32;
	int blockSize = blockDim.x * blockDim.y;
	int paddedWidth = inputWidth + 2 * padding;

	// load filter
	int filterLoadSrcIdx = blockIdx.y * channelGroupSize * filterWidth * filterHeight + threadIdx.x;
	filterData[threadIdx.x] = filter[filterLoadSrcIdx];
	// load rest of the filter value. 9 * 32 in total
	if (threadIdx.x < 9 * 32 - blockSize) {
		filterData[blockSize + threadIdx.x] = filter[blockSize + filterLoadSrcIdx];
	}

	// set left and right padding
	int leftPaddingIdx = threadIdx.x * paddedWidth;
	inputData[leftPaddingIdx] = 0;
	inputData[leftPaddingIdx + paddedWidth - 1] = 0; // right side padding
	__syncthreads();

	// load input
	// for all threads in the same block, use blockIdx.x to find correct batch index, use blockIdx.y to find correct input channel.
	int inputLoadIdxBase = blockIdx.x * inputChannel * inputHeight * inputWidth + blockIdx.y * channelGroupSize * inputHeight * inputWidth;
	int inputLoadSrcIdx = inputLoadIdxBase + threadIdx.x;	// each thread find its own load source.
	int inputLoadDstIdx = (threadIdx.x / inputWidth) * 2 + threadIdx.x + 1;	// each thread find its own load destination.

	inputData[inputLoadDstIdx] = input[inputLoadSrcIdx];
	inputData[inputLoadDstIdx + 32 * 9 * 1] = input[inputLoadSrcIdx + 32 * 7 * 1];
	inputData[inputLoadDstIdx + 32 * 9 * 2] = input[inputLoadSrcIdx + 32 * 7 * 2];
	inputData[inputLoadDstIdx + 32 * 9 * 3] = input[inputLoadSrcIdx + 32 * 7 * 3];
	inputData[inputLoadDstIdx + 32 * 9 * 4] = input[inputLoadSrcIdx + 32 * 7 * 4];
	inputData[inputLoadDstIdx + 32 * 9 * 5] = input[inputLoadSrcIdx + 32 * 7 * 5];
	inputData[inputLoadDstIdx + 32 * 9 * 6] = input[inputLoadSrcIdx + 32 * 7 * 6];
	__syncthreads();

	// convolution
	int outputIdx = blockIdx.x * outputChannel * outputHeight * outputWidth +
		blockIdx.y * channelGroupSize * outputHeight * outputWidth +
		(threadIdx.x / outputWidth) * outputHeight * outputWidth +
		threadIdx.x % outputWidth;

	int inputAccessBase = (threadIdx.x / outputWidth) * paddedWidth * inputHeight + threadIdx.x % outputWidth;
	int filterAccessBase = (threadIdx.x / outputWidth) * filterHeight * filterWidth;
	int inputAccessOffset = 0;

	// 1st row
	// convolve with filter 2 times:
	// 		1. filter's 2nd row (when filter is sliding through the 1st row of input)
	//		2. filter's 1st row (when filter is sliding through the 2nd row of input)
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = filterData[filterAccessBase + 3] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	// 2nd row
	// convolve with filter 3 times:
	//		1. filter's 3rd row (when filter is sliding through the 1st row of input)
	// 		2. filter's 2nd row (when filter is sliding through the 2nd row of input)
	//		3. filter's 1st row (when filter is sliding through the 3rd row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;
	sum2 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// 3rd row
	// convolve with filter 3 times:
	//		1. filter's 3rd row (when filter is sliding through the 2nd row of input)
	// 		2. filter's 2nd row (when filter is sliding through the 3rd row of input)
	//		3. filter's 1st row (when filter is sliding through the 4th row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp0;
	sum0 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// 4th row
	// convolve with filter 3 times:
	//		1. filter's 3rd row (when filter is sliding through the 3rd row of input)
	// 		2. filter's 2nd row (when filter is sliding through the 4th row of input)
	//		3. filter's 1st row (when filter is sliding through the 5th row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum2 * alpha + beta;
	outputIdx += outputWidth;

	// 5th row
	// convolve with filter 3 times:
	//		1. filter's 3rd row (when filter is sliding through the 4th row of input)
	// 		2. filter's 2nd row (when filter is sliding through the 5th row of input)
	//		3. filter's 1st row (when filter is sliding through the 6th row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;
	sum2 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// 6th row
	// convolve with filter 3 times:
	//		1. filter's 3rd row (when filter is sliding through the 5th row of input)
	// 		2. filter's 2nd row (when filter is sliding through the 6th row of input)
	//		3. filter's 1st row (when filter is sliding through the 7th row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp0;
	sum0 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// 7th row
	// convolve with filter 2 times:
	// 		1. filter's 3rd row (when filter is sliding through the 6th row of input)
	//		2. filter's 2nd row (when filter is sliding through the 7th row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;

	output[outputIdx] = sum2 * alpha + beta;
	outputIdx += outputWidth;

	output[outputIdx] = sum0 * alpha + beta;
}

/*
Depthwise Convolution Kernel.

Case: filter 5 x 5, input 7 x 7, stride 1, padding 2

The number of channel must be multiple of 32.
Used in the MobileNet V2 and EfficientNet B0, in case of
	1) 7 x 7 x 1152 -> 7 x 7 x 1152, stride = 1, fitler = 5
*/
__global__ void Filter5x5_Input7x7_Stride1(const float* input, const float* filter, float* output,
	int inputBatchNumber, int inputChannel, int inputHeight, int inputWidth,
	int filterLayerNumber, int filterHeight, int filterWidth,
	int outputBatchNumber, int outputChannel, int outputHeight, int outputWidth,
	int padding, int stride,
	float alpha, float beta) {

	// every 32 channels is a group.
	__shared__ float filterData[32 * 25];	// filter is 5 x 5 = 25
	__shared__ float inputData[32 * 7 * 11]; // original input is 7 x 7, padded to be 11 x 11. ignore up and bottom padding, so 7 x 11

	float inTemp0, inTemp1, inTemp2, inTemp3, inTemp4;
	float sum0, sum1, sum2, sum3, sum4;  // to accumulate the row sum result. rolling recycle.

	int channelGroupSize = 32;
	int paddedWidth = inputWidth + 2 * padding;

	// load filter
	int filterLoadSrcIdx = blockIdx.y * channelGroupSize * filterWidth * filterHeight + threadIdx.x;
	filterData[threadIdx.x] = filter[filterLoadSrcIdx];
	filterData[threadIdx.x + 32 * 7] = filter[filterLoadSrcIdx + 32 * 7];
	filterData[threadIdx.x + 32 * 7 * 2] = filter[filterLoadSrcIdx + 32 * 7 * 2];
	// load rest of the filter value. 25 * 32 in total
	if (threadIdx.x < 25 * 32 - 3 * 32 * 7) {
		filterData[32 * 7 * 3 + threadIdx.x] = filter[32 * 7 * 3 + filterLoadSrcIdx];
	}

	// set left and right padding
	int leftPaddingIdx = threadIdx.x * paddedWidth;
	inputData[leftPaddingIdx] = 0;
	inputData[leftPaddingIdx + 1] = 0;
	inputData[leftPaddingIdx + 9] = 0; // right side padding
	inputData[leftPaddingIdx + 10] = 0; // right side padding
	__syncthreads();

	// load input
	// for all threads in the same block, use blockIdx.x to find correct batch index, use blockIdx.y to find correct input channel.
	int inputLoadIdxBase = blockIdx.x * inputChannel * inputHeight * inputWidth + blockIdx.y * channelGroupSize * inputHeight * inputWidth;
	int inputLoadSrcIdx = inputLoadIdxBase + threadIdx.x;	// each thread find its own load source.
	int inputLoadDstIdx = (threadIdx.x / inputWidth) * 4 + threadIdx.x + 2;	// each thread find its own load destination.

	inputData[inputLoadDstIdx] = input[inputLoadSrcIdx];
	inputData[inputLoadDstIdx + 32 * 11 * 1] = input[inputLoadSrcIdx + 32 * 7 * 1];
	inputData[inputLoadDstIdx + 32 * 11 * 2] = input[inputLoadSrcIdx + 32 * 7 * 2];
	inputData[inputLoadDstIdx + 32 * 11 * 3] = input[inputLoadSrcIdx + 32 * 7 * 3];
	inputData[inputLoadDstIdx + 32 * 11 * 4] = input[inputLoadSrcIdx + 32 * 7 * 4];
	inputData[inputLoadDstIdx + 32 * 11 * 5] = input[inputLoadSrcIdx + 32 * 7 * 5];
	inputData[inputLoadDstIdx + 32 * 11 * 6] = input[inputLoadSrcIdx + 32 * 7 * 6];
	__syncthreads();

	// convolution
	int outputIdx = blockIdx.x * outputChannel * outputHeight * outputWidth +
		blockIdx.y * channelGroupSize * outputHeight * outputWidth +
		(threadIdx.x / outputWidth) * outputHeight * outputWidth +
		threadIdx.x % outputWidth;

	int inputAccessBase = (threadIdx.x / inputWidth) * paddedWidth * inputHeight + threadIdx.x % inputWidth;
	int filterAccessBase = (threadIdx.x / inputWidth) * filterHeight * filterWidth;
	int inputAccessOffset = 0;

	// 1st row
	// convolve with filter 3 times:
	// 		1. filter's 3rd row (when filter is sliding through the 1st row of input) 
	//		2. filter's 2nd row (when filter is sliding through the 2nd row of input) 
	//		3. filter's 1st row (when filter is sliding through the 3rd row of input)
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = filterData[filterAccessBase + 10] * inTemp0;
	sum1 = filterData[filterAccessBase + 5] * inTemp0;
	sum2 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 11] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 12] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 13] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 14] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 9] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp4;

	// 2nd row
	// convolve with filter 4 times:
	//		1. filter's 4th row (when filter is sliding through the 1st row of input)
	// 		2. filter's 3rd row (when filter is sliding through the 2nd row of input) 
	//		3. filter's 2nd row (when filter is sliding through the 3rd row of input) 
	//		3. filter's 1st row (when filter is sliding through the 4th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 15] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 10] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp0;
	sum3 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 16] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 11] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp1;
	sum3 = sum3 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 17] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 12] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp2;
	sum3 = sum3 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 18] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 13] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp3;
	sum3 = sum3 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 19] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 14] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 9] * inTemp4;
	sum3 = sum3 + filterData[filterAccessBase + 4] * inTemp4;

	// 3rd row
	// convolve with filter 5 times:
	//		1. filter's 5th row (when filter is sliding through the 1st row of input)
	// 		2. filter's 4th row (when filter is sliding through the 2nd row of input) 
	//		3. filter's 3rd row (when filter is sliding through the 3rd row of input) 
	//		4. filter's 2nd row (when filter is sliding through the 4th row of input) 
	//		5. filter's 1st row (when filter is sliding through the 5th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 20] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 15] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 10] * inTemp0;
	sum3 = sum3 + filterData[filterAccessBase + 5] * inTemp0;
	sum4 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 21] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 16] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 11] * inTemp1;
	sum3 = sum3 + filterData[filterAccessBase + 6] * inTemp1;
	sum4 = sum4 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 22] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 17] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 12] * inTemp2;
	sum3 = sum3 + filterData[filterAccessBase + 7] * inTemp2;
	sum4 = sum4 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 23] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 18] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 13] * inTemp3;
	sum3 = sum3 + filterData[filterAccessBase + 8] * inTemp3;
	sum4 = sum4 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 24] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 19] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 14] * inTemp4;
	sum3 = sum3 + filterData[filterAccessBase + 9] * inTemp4;
	sum4 = sum4 + filterData[filterAccessBase + 4] * inTemp4;
	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// 4th row
	// convolve with filter 5 times:
	//		1. filter's 5th row (when filter is sliding through the 2nd row of input)
	// 		2. filter's 4th row (when filter is sliding through the 3rd row of input) 
	//		3. filter's 3rd row (when filter is sliding through the 4th row of input) 
	//		4. filter's 2nd row (when filter is sliding through the 5th row of input) 
	//		5. filter's 1st row (when filter is sliding through the 6th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 20] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 15] * inTemp0;
	sum3 = sum3 + filterData[filterAccessBase + 10] * inTemp0;
	sum4 = sum4 + filterData[filterAccessBase + 5] * inTemp0;
	sum0 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 21] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 16] * inTemp1;
	sum3 = sum3 + filterData[filterAccessBase + 11] * inTemp1;
	sum4 = sum4 + filterData[filterAccessBase + 6] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 22] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 17] * inTemp2;
	sum3 = sum3 + filterData[filterAccessBase + 12] * inTemp2;
	sum4 = sum4 + filterData[filterAccessBase + 7] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 23] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 18] * inTemp3;
	sum3 = sum3 + filterData[filterAccessBase + 13] * inTemp3;
	sum4 = sum4 + filterData[filterAccessBase + 8] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 24] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 19] * inTemp4;
	sum3 = sum3 + filterData[filterAccessBase + 14] * inTemp4;
	sum4 = sum4 + filterData[filterAccessBase + 9] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp4;
	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// 5th row
	// convolve with filter 5 times:
	//		1. filter's 5th row (when filter is sliding through the 3rd row of input)
	// 		2. filter's 4th row (when filter is sliding through the 4th row of input) 
	//		3. filter's 3rd row (when filter is sliding through the 5th row of input) 
	// 		4. filter's 2nd row (when filter is sliding through the 6th row of input) 
	//		5. filter's 1st row (when filter is sliding through the 7th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 20] * inTemp0;
	sum3 = sum3 + filterData[filterAccessBase + 15] * inTemp0;
	sum4 = sum4 + filterData[filterAccessBase + 10] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 21] * inTemp1;
	sum3 = sum3 + filterData[filterAccessBase + 16] * inTemp1;
	sum4 = sum4 + filterData[filterAccessBase + 11] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 22] * inTemp2;
	sum3 = sum3 + filterData[filterAccessBase + 17] * inTemp2;
	sum4 = sum4 + filterData[filterAccessBase + 12] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 23] * inTemp3;
	sum3 = sum3 + filterData[filterAccessBase + 18] * inTemp3;
	sum4 = sum4 + filterData[filterAccessBase + 13] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 24] * inTemp4;
	sum3 = sum3 + filterData[filterAccessBase + 19] * inTemp4;
	sum4 = sum4 + filterData[filterAccessBase + 14] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 9] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp4;

	output[outputIdx] = sum2 * alpha + beta;
	outputIdx += outputWidth;

	// 6th row
	// convolve with filter 4 times:
	//		1. filter's 5th row (when filter is sliding through the 4th row of input)
	//		2. filter's 4th row (when filter is sliding through the 5th row of input)
	// 		3. filter's 3rd row (when filter is sliding through the 6th row of input) 
	//		4. filter's 2nd row (when filter is sliding through the 7th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum3 = sum3 + filterData[filterAccessBase + 20] * inTemp0;
	sum4 = sum4 + filterData[filterAccessBase + 15] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 10] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum3 = sum3 + filterData[filterAccessBase + 21] * inTemp1;
	sum4 = sum4 + filterData[filterAccessBase + 16] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 11] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum3 = sum3 + filterData[filterAccessBase + 22] * inTemp2;
	sum4 = sum4 + filterData[filterAccessBase + 17] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 12] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum3 = sum3 + filterData[filterAccessBase + 23] * inTemp3;
	sum4 = sum4 + filterData[filterAccessBase + 18] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 13] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum3 = sum3 + filterData[filterAccessBase + 24] * inTemp4;
	sum4 = sum4 + filterData[filterAccessBase + 19] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 14] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 9] * inTemp4;

	output[outputIdx] = sum3 * alpha + beta;
	outputIdx += outputWidth;

	// 7th row
	// convolve with filter 3 times:
	// 		1. filter's 5th row (when filter is sliding through the 5th row of input) 
	//		2. filter's 4th row (when filter is sliding through the 6th row of input) 
	//		3. filter's 3rd row (when filter is sliding through the 7th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum4 = sum4 + filterData[filterAccessBase + 20] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 15] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 10] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum4 = sum4 + filterData[filterAccessBase + 21] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 16] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 11] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum4 = sum4 + filterData[filterAccessBase + 22] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 17] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 12] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum4 = sum4 + filterData[filterAccessBase + 23] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 18] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 13] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum4 = sum4 + filterData[filterAccessBase + 24] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 19] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 14] * inTemp4;

	output[outputIdx] = sum4 * alpha + beta;
	outputIdx += outputWidth;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	output[outputIdx] = sum1 * alpha + beta;
}

/*
Depthwise Convolution Kernel.

Case: filter 3 x 3, input 14 x 14, stride 1, padding 1

The number of channel must be multiple of 16.
Used in the MobileNet V2 and EfficientNet B0, in case of
	1)	14 x 14 x 384 -> 14 x 14 x 384, stride = 1, filter = 3
	2)	14 x 14 x 480 -> 14 x 14 x 480, stride = 1, filter = 3
	3)	14 x 14 x 576 -> 14 x 14 x 576, stride = 1, filter = 3
*/
__global__ void Filter3x3_Input14x14_Stride1(const float* input, const float* filter, float* output,
	int inputBatchNumber, int inputChannel, int inputHeight, int inputWidth,
	int filterLayerNumber, int filterHeight, int filterWidth,
	int outputBatchNumber, int outputChannel, int outputHeight, int outputWidth,
	int padding, int stride,
	float alpha, float beta) {

	// every 16 channels is a group.
	__shared__ float filterData[16 * 9];	// filter is 3 x 3 = 9
	__shared__ float inputData[16 * 14 * 16]; // original input is 14 x 14, padded to be 16 x 16. ignore up and bottom padding, so 14 x 16

	float inTemp0, inTemp1, inTemp2;
	float sum0, sum1, sum2;  // to accumulate the row sum result. rolling recycle.

	int channelGroupSize = 16;
	int paddedWidth = inputWidth + 2 * padding;

	// load filter
	int filterLoadSrcIdx = blockIdx.y * channelGroupSize * filterWidth * filterHeight + threadIdx.x;
	if (threadIdx.x < 16 * 9) {
		filterData[threadIdx.x] = filter[filterLoadSrcIdx];
	}

	// set left and right padding
	int leftPaddingIdx = threadIdx.x * paddedWidth;
	inputData[leftPaddingIdx] = 0;
	inputData[leftPaddingIdx + 15] = 0; // right side padding

	__syncthreads();

	// load input
	// for all threads in the same block, use blockIdx.x to find correct batch index, use blockIdx.y to find correct input channel.
	int inputLoadIdxBase = blockIdx.x * inputChannel * inputHeight * inputWidth + blockIdx.y * channelGroupSize * inputHeight * inputWidth;
	int inputLoadSrcIdx = inputLoadIdxBase + threadIdx.x;	// each thread find its own load source.
	int inputLoadDstIdx = (threadIdx.x / inputWidth) * 2 + threadIdx.x + 1;	// each thread find its own load destination.

	inputData[inputLoadDstIdx] = input[inputLoadSrcIdx];
	inputData[inputLoadDstIdx + 16 * 16 * 1] = input[inputLoadSrcIdx + 16 * 14 * 1];
	inputData[inputLoadDstIdx + 16 * 16 * 2] = input[inputLoadSrcIdx + 16 * 14 * 2];
	inputData[inputLoadDstIdx + 16 * 16 * 3] = input[inputLoadSrcIdx + 16 * 14 * 3];
	inputData[inputLoadDstIdx + 16 * 16 * 4] = input[inputLoadSrcIdx + 16 * 14 * 4];
	inputData[inputLoadDstIdx + 16 * 16 * 5] = input[inputLoadSrcIdx + 16 * 14 * 5];
	inputData[inputLoadDstIdx + 16 * 16 * 6] = input[inputLoadSrcIdx + 16 * 14 * 6];
	inputData[inputLoadDstIdx + 16 * 16 * 7] = input[inputLoadSrcIdx + 16 * 14 * 7];
	inputData[inputLoadDstIdx + 16 * 16 * 8] = input[inputLoadSrcIdx + 16 * 14 * 8];
	inputData[inputLoadDstIdx + 16 * 16 * 9] = input[inputLoadSrcIdx + 16 * 14 * 9];
	inputData[inputLoadDstIdx + 16 * 16 * 10] = input[inputLoadSrcIdx + 16 * 14 * 10];
	inputData[inputLoadDstIdx + 16 * 16 * 11] = input[inputLoadSrcIdx + 16 * 14 * 11];
	inputData[inputLoadDstIdx + 16 * 16 * 12] = input[inputLoadSrcIdx + 16 * 14 * 12];
	inputData[inputLoadDstIdx + 16 * 16 * 13] = input[inputLoadSrcIdx + 16 * 14 * 13];
	__syncthreads();

	// convolution
	int outputIdx = blockIdx.x * outputChannel * outputHeight * outputWidth +
		blockIdx.y * channelGroupSize * outputHeight * outputWidth +
		(threadIdx.x / outputWidth) * outputHeight * outputWidth +
		threadIdx.x % outputWidth;

	int inputAccessBase = (threadIdx.x / outputWidth) * paddedWidth * inputHeight + threadIdx.x % inputWidth;
	int filterAccessBase = (threadIdx.x / inputWidth) * filterHeight * filterWidth;
	int inputAccessOffset = 0;

	// 1st row
	// convolve with filter 2 times:
	// 		1. filter's 2nd row (when filter is sliding through the 1st row of input) 
	//		2. filter's 1st row (when filter is sliding through the 2nd row of input) 
	inTemp0 = inputData[inputAccessBase];
	sum0 = filterData[filterAccessBase + 3] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1];
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2];
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	// 2nd row
	// convolve with filter 3 times:
	//		1. filter's 3rd row (when filter is sliding through the 1st row of input)
	// 		2. filter's 2nd row (when filter is sliding through the 2nd row of input) 
	//		3. filter's 1st row (when filter is sliding through the 3rd row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;
	sum2 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// 3rd row
	// convolve with filter 3 times:
	//		1. filter's 3rd row (when filter is sliding through the 2nd row of input)
	// 		2. filter's 2nd row (when filter is sliding through the 3rd row of input) 
	//		3. filter's 1st row (when filter is sliding through the 4th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp0;
	sum0 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// 4th row
	// convolve with filter 3 times:
	//		1. filter's 3rd row (when filter is sliding through the 3rd row of input)
	// 		2. filter's 2nd row (when filter is sliding through the 4th row of input) 
	//		3. filter's 1st row (when filter is sliding through the 5th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum2 * alpha + beta;
	outputIdx += outputWidth;

	// 5th row
	// convolve with filter 3 times:
	//		1. filter's 3rd row (when filter is sliding through the 4th row of input)
	// 		2. filter's 2nd row (when filter is sliding through the 5th row of input) 
	//		3. filter's 1st row (when filter is sliding through the 6th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;
	sum2 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// 6th row
	// convolve with filter 3 times:
	//		1. filter's 3rd row (when filter is sliding through the 5th row of input)
	// 		2. filter's 2nd row (when filter is sliding through the 6th row of input) 
	//		3. filter's 1st row (when filter is sliding through the 7th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp0;
	sum0 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// 7th row
	// convolve with filter 3 times:
	// 		1. filter's 3rd row (when filter is sliding through the 6th row of input) 
	//		2. filter's 2nd row (when filter is sliding through the 7th row of input) 
	//		3. filter's 1st row (when filter is sliding through the 8th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum2 * alpha + beta;
	outputIdx += outputWidth;

	// 8th row
	// convolve with filter 3 times:
	// 		1. filter's 3rd row (when filter is sliding through the 7th row of input) 
	//		2. filter's 2nd row (when filter is sliding through the 8th row of input) 
	//		3. filter's 1st row (when filter is sliding through the 9th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;
	sum2 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// 9th row
	// convolve with filter 3 times:
	// 		1. filter's 3rd row (when filter is sliding through the 8th row of input) 
	//		2. filter's 2nd row (when filter is sliding through the 9th row of input) 
	//		3. filter's 1st row (when filter is sliding through the 10th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp0;
	sum0 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// 10th row
	// convolve with filter 3 times:
	// 		1. filter's 3rd row (when filter is sliding through the 9th row of input) 
	//		2. filter's 2nd row (when filter is sliding through the 10th row of input) 
	//		3. filter's 1st row (when filter is sliding through the 11th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum2 * alpha + beta;
	outputIdx += outputWidth;

	// 11st row
	// convolve with filter 3 times:
	// 		1. filter's 3rd row (when filter is sliding through the 10th row of input) 
	//		2. filter's 2nd row (when filter is sliding through the 11th row of input) 
	//		3. filter's 1st row (when filter is sliding through the 12th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;
	sum2 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// 12nd row
	// convolve with filter 3 times:
	// 		1. filter's 3rd row (when filter is sliding through the 11th row of input) 
	//		2. filter's 2nd row (when filter is sliding through the 12th row of input) 
	//		3. filter's 1st row (when filter is sliding through the 13th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp0;
	sum0 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// 13rd row
	// convolve with filter 3 times:
	// 		1. filter's 3rd row (when filter is sliding through the 12th row of input) 
	//		2. filter's 2nd row (when filter is sliding through the 13th row of input) 
	//		3. filter's 1st row (when filter is sliding through the 14th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum2 * alpha + beta;
	outputIdx += outputWidth;

	// 14th row
	// convolve with filter 2 times:
	//		1. filter's 2nd row (when filter is sliding through the 13th row of input) 
	//		2. filter's 1st row (when filter is sliding through the 14th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	output[outputIdx] = sum1 * alpha + beta;
}

/*
Depthwise Convolution Kernel.

Case: filter 3 x 3, input 14 x 14, stride 2, padding 1

The number of channel must be multiple of 32.
Used in the MobileNet V2 and EfficientNet B0, in case of
	1)	14 x 14 x 576 -> 14 x 14 x 576, stride = 2, filter = 3
*/
__global__ void Filter3x3_Input14x14_Stride2(const float* input, const float* filter, float* output,
	int inputBatchNumber, int inputChannel, int inputHeight, int inputWidth,
	int filterLayerNumber, int filterHeight, int filterWidth,
	int outputBatchNumber, int outputChannel, int outputHeight, int outputWidth,
	int padding, int stride,
	float alpha, float beta) {

	// every 32 channels is a group.
	__shared__ float filterData[32 * 9];	// filter is 3 x 3 = 9
	__shared__ float inputData[32 * 14 * 16]; // original input is 14 x 14, padded to be 16 x 16. ignore up and bottom padding, so 14 x 16

	float inTemp0, inTemp1, inTemp2;
	float sum0, sum1;  // to accumulate the row sum result. rolling recycle.

	int channelGroupSize = 32;
	int paddedWidth = inputWidth + 2 * padding;

	// load filter
	int filterLoadSrcIdx = blockIdx.y * channelGroupSize * filterWidth * filterHeight + threadIdx.x;
	filterData[threadIdx.x] = filter[filterLoadSrcIdx];
	// load rest of the filter value. 9 * 32 in total
	if (threadIdx.x < 9 * 32 - 7 * 32) {
		filterData[7 * 32 + threadIdx.x] = filter[7 * 32 + filterLoadSrcIdx];
	}

	// set left and right padding
	int leftPaddingIdx = threadIdx.x * paddedWidth;
	inputData[leftPaddingIdx] = 0;		// left padding upper half
	inputData[leftPaddingIdx + 16 * inputHeight * paddedWidth] = 0; // left padding bottom half
	inputData[leftPaddingIdx + 15] = 0; // right padding upper half
	inputData[leftPaddingIdx + 16 * inputHeight * paddedWidth + 15] = 0; // right padding bottom half

	__syncthreads();

	// load input
	// for all threads in the same block, use blockIdx.x to find correct batch index, use blockIdx.y to find correct input channel.
	int inputLoadIdxBase = blockIdx.x * inputChannel * inputHeight * inputWidth + blockIdx.y * channelGroupSize * inputHeight * inputWidth;
	int inputLoadSrcIdx = inputLoadIdxBase + threadIdx.x;	// each thread find its own load source.
	int inputLoadDstIdx = (threadIdx.x / inputWidth) * 2 + threadIdx.x + 1;	// each thread find its own load destination.

	inputData[inputLoadDstIdx] = input[inputLoadSrcIdx];
	inputData[inputLoadDstIdx + 16 * 16 * 1] = input[inputLoadSrcIdx + 16 * 14 * 1];
	inputData[inputLoadDstIdx + 16 * 16 * 2] = input[inputLoadSrcIdx + 16 * 14 * 2];
	inputData[inputLoadDstIdx + 16 * 16 * 3] = input[inputLoadSrcIdx + 16 * 14 * 3];
	inputData[inputLoadDstIdx + 16 * 16 * 4] = input[inputLoadSrcIdx + 16 * 14 * 4];
	inputData[inputLoadDstIdx + 16 * 16 * 5] = input[inputLoadSrcIdx + 16 * 14 * 5];
	inputData[inputLoadDstIdx + 16 * 16 * 6] = input[inputLoadSrcIdx + 16 * 14 * 6];
	inputData[inputLoadDstIdx + 16 * 16 * 7] = input[inputLoadSrcIdx + 16 * 14 * 7];
	inputData[inputLoadDstIdx + 16 * 16 * 8] = input[inputLoadSrcIdx + 16 * 14 * 8];
	inputData[inputLoadDstIdx + 16 * 16 * 9] = input[inputLoadSrcIdx + 16 * 14 * 9];
	inputData[inputLoadDstIdx + 16 * 16 * 10] = input[inputLoadSrcIdx + 16 * 14 * 10];
	inputData[inputLoadDstIdx + 16 * 16 * 11] = input[inputLoadSrcIdx + 16 * 14 * 11];
	inputData[inputLoadDstIdx + 16 * 16 * 12] = input[inputLoadSrcIdx + 16 * 14 * 12];
	inputData[inputLoadDstIdx + 16 * 16 * 13] = input[inputLoadSrcIdx + 16 * 14 * 13];
	inputData[inputLoadDstIdx + 16 * 16 * 14] = input[inputLoadSrcIdx + 16 * 14 * 14];
	inputData[inputLoadDstIdx + 16 * 16 * 15] = input[inputLoadSrcIdx + 16 * 14 * 15];
	inputData[inputLoadDstIdx + 16 * 16 * 16] = input[inputLoadSrcIdx + 16 * 14 * 16];
	inputData[inputLoadDstIdx + 16 * 16 * 17] = input[inputLoadSrcIdx + 16 * 14 * 17];
	inputData[inputLoadDstIdx + 16 * 16 * 18] = input[inputLoadSrcIdx + 16 * 14 * 18];
	inputData[inputLoadDstIdx + 16 * 16 * 19] = input[inputLoadSrcIdx + 16 * 14 * 19];
	inputData[inputLoadDstIdx + 16 * 16 * 20] = input[inputLoadSrcIdx + 16 * 14 * 20];
	inputData[inputLoadDstIdx + 16 * 16 * 21] = input[inputLoadSrcIdx + 16 * 14 * 21];
	inputData[inputLoadDstIdx + 16 * 16 * 22] = input[inputLoadSrcIdx + 16 * 14 * 22];
	inputData[inputLoadDstIdx + 16 * 16 * 23] = input[inputLoadSrcIdx + 16 * 14 * 23];
	inputData[inputLoadDstIdx + 16 * 16 * 24] = input[inputLoadSrcIdx + 16 * 14 * 24];
	inputData[inputLoadDstIdx + 16 * 16 * 25] = input[inputLoadSrcIdx + 16 * 14 * 25];
	inputData[inputLoadDstIdx + 16 * 16 * 26] = input[inputLoadSrcIdx + 16 * 14 * 26];
	inputData[inputLoadDstIdx + 16 * 16 * 27] = input[inputLoadSrcIdx + 16 * 14 * 27];
	__syncthreads();

	// convolution
	int outputIdx = blockIdx.x * outputChannel * outputHeight * outputWidth +
		blockIdx.y * channelGroupSize * outputHeight * outputWidth +
		(threadIdx.x / outputWidth) * outputHeight * outputWidth + threadIdx.x % outputWidth;

	int inputAccessBase = (threadIdx.x / outputWidth) * paddedWidth * inputHeight + threadIdx.x % outputWidth * 2;
	int filterAccessBase = (threadIdx.x / outputWidth) * filterHeight * filterWidth;
	int inputAccessOffset = 0;

	// 1st row
	// convolve with filter 1 time:
	// 		1. filter's 2nd row (when filter is sliding through the 1st row of input) 
	inTemp0 = inputData[inputAccessBase];
	sum0 = filterData[filterAccessBase + 3] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1];
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2];
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;

	// 2nd row
	// convolve with filter 2 times:
	//		1. filter's 3rd row (when filter is sliding through the 1st row of input)
	//		2. filter's 1st row (when filter is sliding through the 3rd row of input) 
	inputAccessOffset += paddedWidth;

	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// 3rd row
	// convolve with filter 1 time:
	// 		1. filter's 2nd row (when filter is sliding through the 3rd row of input) 
	inputAccessOffset += paddedWidth;

	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;

	// 4th row
	// convolve with filter 2 times:
	//		1. filter's 3rd row (when filter is sliding through the 3rd row of input)
	//		2. filter's 1st row (when filter is sliding through the 5th row of input) 
	inputAccessOffset += paddedWidth;

	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;
	sum0 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// 5th row
	// convolve with filter 1 time:
	// 		1. filter's 2nd row (when filter is sliding through the 5th row of input)
	inputAccessOffset += paddedWidth;

	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;

	// 6th row
	// convolve with filter 2 times:
	//		1. filter's 3rd row (when filter is sliding through the 5th row of input)
	//		2. filter's 1st row (when filter is sliding through the 7th row of input) 
	inputAccessOffset += paddedWidth;

	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// 7th row
	// convolve with filter 1 time:
	//		1. filter's 2nd row (when filter is sliding through the 7th row of input) 
	inputAccessOffset += paddedWidth;

	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;

	// 8th row
	// convolve with filter 2 times:
	//		1. filter's 3rd row (when filter is sliding through the 7th row of input) 
	//		2. filter's 1nd row (when filter is sliding through the 9th row of input) 
	inputAccessOffset += paddedWidth;

	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;
	sum0 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// 9th row
	// convolve with filter 1 time:
	//		1. filter's 2nd row (when filter is sliding through the 9th row of input) 
	inputAccessOffset += paddedWidth;

	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;

	// 10th row
	// convolve with filter 2 times:
	//		1. filter's 3rd row (when filter is sliding through the 9th row of input) 
	//		2. filter's 1nd row (when filter is sliding through the 11th row of input) 
	inputAccessOffset += paddedWidth;

	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// 11st row
	// convolve with filter 1 time:
	//		1. filter's 2nd row (when filter is sliding through the 11th row of input) 
	inputAccessOffset += paddedWidth;

	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;

	// 12nd row
	// convolve with filter 2 times:
	//		1. filter's 3rd row (when filter is sliding through the 11th row of input) 
	//		2. filter's 1nd row (when filter is sliding through the 13th row of input) 
	inputAccessOffset += paddedWidth;

	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;
	sum0 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// 13rd row
	// convolve with filter 1 time:
	//		1. filter's 2nd row (when filter is sliding through the 13th row of input) 
	inputAccessOffset += paddedWidth;

	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;

	// 14th row
	// convolve with filter 1 time:
	//		1. filter's 3rd row (when filter is sliding through the 13th row of input) 
	inputAccessOffset += paddedWidth;

	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;

	output[outputIdx] = sum0 * alpha + beta;
}

/*
Depthwise Convolution Kernel.

Case: filter 5 x 5, input 14 x 14, stride 1, padding 2

The number of channel must be multiple of 16.
Used in the MobileNet V2 and EfficientNet B0, in case of.
	1)	14 x 14 x 480 -> 14 x 14 x 480, stride = 1, filter = 5
	2)	14 x 14 x 672 -> 14 x 14 x 672, stride = 1, filter = 5

*/
__global__ void Filter5x5_Input14x14_Stride1(const float* input, const float* filter, float* output,
	int inputBatchNumber, int inputChannel, int inputHeight, int inputWidth,
	int filterLayerNumber, int filterHeight, int filterWidth,
	int outputBatchNumber, int outputChannel, int outputHeight, int outputWidth,
	int padding, int stride,
	float alpha, float beta) {

	// every 16 channels is a group.
	__shared__ float filterData[16 * 25];	// filter is 5 x 5 = 25
	__shared__ float inputData[16 * 14 * 18]; // original input is 14 x 14, padded to be 18 x 18. ignore up and bottom padding, so 14 x 18

	float inTemp0, inTemp1, inTemp2, inTemp3, inTemp4;
	float sum0, sum1, sum2, sum3, sum4;  // to accumulate the row sum result. rolling recycle.

	int channelGroupSize = 16;
	// int blockSize = blockDim.x * blockDim.y;
	int paddedWidth = inputWidth + 2 * padding;

	// load filter
	int filterLoadSrcIdx = blockIdx.y * channelGroupSize * filterWidth * filterHeight + threadIdx.x;
	if (threadIdx.x < 8 * 25) {
		filterData[threadIdx.x] = filter[filterLoadSrcIdx];
		filterData[threadIdx.x + 8 * 25] = filter[filterLoadSrcIdx + 8 * 25];
	}

	// set left and right padding
	int leftPaddingIdx = threadIdx.x * paddedWidth;
	inputData[leftPaddingIdx] = 0;
	inputData[leftPaddingIdx + 1] = 0;
	inputData[leftPaddingIdx + 16] = 0; // right side padding
	inputData[leftPaddingIdx + 17] = 0; // right side padding

	__syncthreads();

	// load input
	// for all threads in the same block, use blockIdx.x to find correct batch index, use blockIdx.y to find correct input channel.
	int inputLoadIdxBase = blockIdx.x * inputChannel * inputHeight * inputWidth + blockIdx.y * channelGroupSize * inputHeight * inputWidth;
	int inputLoadSrcIdx = inputLoadIdxBase + threadIdx.x;	// each thread find its own load source.
	int inputLoadDstIdx = (threadIdx.x / inputWidth) * 4 + threadIdx.x + 2;	// each thread find its own load destination.

	inputData[inputLoadDstIdx] = input[inputLoadSrcIdx];
	inputData[inputLoadDstIdx + 16 * 18 * 1] = input[inputLoadSrcIdx + 16 * 14 * 1];
	inputData[inputLoadDstIdx + 16 * 18 * 2] = input[inputLoadSrcIdx + 16 * 14 * 2];
	inputData[inputLoadDstIdx + 16 * 18 * 3] = input[inputLoadSrcIdx + 16 * 14 * 3];
	inputData[inputLoadDstIdx + 16 * 18 * 4] = input[inputLoadSrcIdx + 16 * 14 * 4];
	inputData[inputLoadDstIdx + 16 * 18 * 5] = input[inputLoadSrcIdx + 16 * 14 * 5];
	inputData[inputLoadDstIdx + 16 * 18 * 6] = input[inputLoadSrcIdx + 16 * 14 * 6];
	inputData[inputLoadDstIdx + 16 * 18 * 7] = input[inputLoadSrcIdx + 16 * 14 * 7];
	inputData[inputLoadDstIdx + 16 * 18 * 8] = input[inputLoadSrcIdx + 16 * 14 * 8];
	inputData[inputLoadDstIdx + 16 * 18 * 9] = input[inputLoadSrcIdx + 16 * 14 * 9];
	inputData[inputLoadDstIdx + 16 * 18 * 10] = input[inputLoadSrcIdx + 16 * 14 * 10];
	inputData[inputLoadDstIdx + 16 * 18 * 11] = input[inputLoadSrcIdx + 16 * 14 * 11];
	inputData[inputLoadDstIdx + 16 * 18 * 12] = input[inputLoadSrcIdx + 16 * 14 * 12];
	inputData[inputLoadDstIdx + 16 * 18 * 13] = input[inputLoadSrcIdx + 16 * 14 * 13];
	__syncthreads();

	// convolution
	int outputIdx = blockIdx.x * outputChannel * outputHeight * outputWidth +
		blockIdx.y * channelGroupSize * outputHeight * outputWidth +
		(threadIdx.x / outputWidth) * outputHeight * outputWidth +
		threadIdx.x % outputWidth;

	int inputAccessBase = (threadIdx.x / outputWidth) * paddedWidth * inputHeight + threadIdx.x % outputWidth;
	int filterAccessBase = (threadIdx.x / inputWidth) * filterHeight * filterWidth;
	int inputAccessOffset = 0;

	// 1st row
	// convolve with filter 3 times:
	// 		1. filter's 3rd row (when filter is sliding through the 1st row of input) 
	//		2. filter's 2nd row (when filter is sliding through the 2nd row of input) 
	//		3. filter's 1st row (when filter is sliding through the 3rd row of input)
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = filterData[filterAccessBase + 10] * inTemp0;
	sum1 = filterData[filterAccessBase + 5] * inTemp0;
	sum2 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 11] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 12] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 13] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 14] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 9] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp4;

	// 2nd row
	// convolve with filter 4 times:
	//		1. filter's 4th row (when filter is sliding through the 1st row of input)
	// 		2. filter's 3rd row (when filter is sliding through the 2nd row of input) 
	//		3. filter's 2nd row (when filter is sliding through the 3rd row of input) 
	//		3. filter's 1st row (when filter is sliding through the 4th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 15] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 10] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp0;
	sum3 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 16] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 11] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp1;
	sum3 = sum3 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 17] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 12] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp2;
	sum3 = sum3 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 18] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 13] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp3;
	sum3 = sum3 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 19] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 14] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 9] * inTemp4;
	sum3 = sum3 + filterData[filterAccessBase + 4] * inTemp4;

#pragma unroll
	for (int i = 0; i < 2; i++) {
		// 3rd row, 8th row
		// convolve with filter 5 times:
		//		1. filter's 5th row (when filter is sliding through the 1st row of input)
		// 		2. filter's 4th row (when filter is sliding through the 2nd row of input) 
		//		3. filter's 3rd row (when filter is sliding through the 3rd row of input) 
		//		4. filter's 2nd row (when filter is sliding through the 4th row of input) 
		//		5. filter's 1st row (when filter is sliding through the 5th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 20] * inTemp0;
		sum1 = sum1 + filterData[filterAccessBase + 15] * inTemp0;
		sum2 = sum2 + filterData[filterAccessBase + 10] * inTemp0;
		sum3 = sum3 + filterData[filterAccessBase + 5] * inTemp0;
		sum4 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 21] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 16] * inTemp1;
		sum2 = sum2 + filterData[filterAccessBase + 11] * inTemp1;
		sum3 = sum3 + filterData[filterAccessBase + 6] * inTemp1;
		sum4 = sum4 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 22] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 17] * inTemp2;
		sum2 = sum2 + filterData[filterAccessBase + 12] * inTemp2;
		sum3 = sum3 + filterData[filterAccessBase + 7] * inTemp2;
		sum4 = sum4 + filterData[filterAccessBase + 2] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 23] * inTemp3;
		sum1 = sum1 + filterData[filterAccessBase + 18] * inTemp3;
		sum2 = sum2 + filterData[filterAccessBase + 13] * inTemp3;
		sum3 = sum3 + filterData[filterAccessBase + 8] * inTemp3;
		sum4 = sum4 + filterData[filterAccessBase + 3] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 24] * inTemp4;
		sum1 = sum1 + filterData[filterAccessBase + 19] * inTemp4;
		sum2 = sum2 + filterData[filterAccessBase + 14] * inTemp4;
		sum3 = sum3 + filterData[filterAccessBase + 9] * inTemp4;
		sum4 = sum4 + filterData[filterAccessBase + 4] * inTemp4;
		output[outputIdx] = sum0 * alpha + beta;
		outputIdx += outputWidth;

		// 4th row, 9th row
		// convolve with filter 5 times:
		//		1. filter's 5th row (when filter is sliding through the 2nd row of input)
		// 		2. filter's 4th row (when filter is sliding through the 3rd row of input) 
		//		3. filter's 3rd row (when filter is sliding through the 4th row of input) 
		//		4. filter's 2nd row (when filter is sliding through the 5th row of input) 
		//		5. filter's 1st row (when filter is sliding through the 6th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 20] * inTemp0;
		sum2 = sum2 + filterData[filterAccessBase + 15] * inTemp0;
		sum3 = sum3 + filterData[filterAccessBase + 10] * inTemp0;
		sum4 = sum4 + filterData[filterAccessBase + 5] * inTemp0;
		sum0 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 21] * inTemp1;
		sum2 = sum2 + filterData[filterAccessBase + 16] * inTemp1;
		sum3 = sum3 + filterData[filterAccessBase + 11] * inTemp1;
		sum4 = sum4 + filterData[filterAccessBase + 6] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 22] * inTemp2;
		sum2 = sum2 + filterData[filterAccessBase + 17] * inTemp2;
		sum3 = sum3 + filterData[filterAccessBase + 12] * inTemp2;
		sum4 = sum4 + filterData[filterAccessBase + 7] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 23] * inTemp3;
		sum2 = sum2 + filterData[filterAccessBase + 18] * inTemp3;
		sum3 = sum3 + filterData[filterAccessBase + 13] * inTemp3;
		sum4 = sum4 + filterData[filterAccessBase + 8] * inTemp3;
		sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 24] * inTemp4;
		sum2 = sum2 + filterData[filterAccessBase + 19] * inTemp4;
		sum3 = sum3 + filterData[filterAccessBase + 14] * inTemp4;
		sum4 = sum4 + filterData[filterAccessBase + 9] * inTemp4;
		sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp4;
		output[outputIdx] = sum1 * alpha + beta;
		outputIdx += outputWidth;

		// 5th row, 10th row
		// convolve with filter 5 times:
		//		1. filter's 5th row (when filter is sliding through the 3rd row of input)
		// 		2. filter's 4th row (when filter is sliding through the 4th row of input) 
		//		3. filter's 3rd row (when filter is sliding through the 5th row of input) 
		// 		4. filter's 2nd row (when filter is sliding through the 6th row of input) 
		//		5. filter's 1st row (when filter is sliding through the 7th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 20] * inTemp0;
		sum3 = sum3 + filterData[filterAccessBase + 15] * inTemp0;
		sum4 = sum4 + filterData[filterAccessBase + 10] * inTemp0;
		sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp0;
		sum1 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 21] * inTemp1;
		sum3 = sum3 + filterData[filterAccessBase + 16] * inTemp1;
		sum4 = sum4 + filterData[filterAccessBase + 11] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 22] * inTemp2;
		sum3 = sum3 + filterData[filterAccessBase + 17] * inTemp2;
		sum4 = sum4 + filterData[filterAccessBase + 12] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 23] * inTemp3;
		sum3 = sum3 + filterData[filterAccessBase + 18] * inTemp3;
		sum4 = sum4 + filterData[filterAccessBase + 13] * inTemp3;
		sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp3;
		sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 24] * inTemp4;
		sum3 = sum3 + filterData[filterAccessBase + 19] * inTemp4;
		sum4 = sum4 + filterData[filterAccessBase + 14] * inTemp4;
		sum0 = sum0 + filterData[filterAccessBase + 9] * inTemp4;
		sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp4;

		output[outputIdx] = sum2 * alpha + beta;
		outputIdx += outputWidth;

		// 6th row, 11th row
		// convolve with filter 5 times:
		//		1. filter's 5th row (when filter is sliding through the 4th row of input)
		// 		2. filter's 4th row (when filter is sliding through the 5th row of input) 
		//		3. filter's 3rd row (when filter is sliding through the 6th row of input) 
		// 		4. filter's 2nd row (when filter is sliding through the 7th row of input) 
		//		5. filter's 1st row (when filter is sliding through the 8th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum3 = sum3 + filterData[filterAccessBase + 20] * inTemp0;
		sum4 = sum4 + filterData[filterAccessBase + 15] * inTemp0;
		sum0 = sum0 + filterData[filterAccessBase + 10] * inTemp0;
		sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp0;
		sum2 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum3 = sum3 + filterData[filterAccessBase + 21] * inTemp1;
		sum4 = sum4 + filterData[filterAccessBase + 16] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 11] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp1;
		sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum3 = sum3 + filterData[filterAccessBase + 22] * inTemp2;
		sum4 = sum4 + filterData[filterAccessBase + 17] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 12] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp2;
		sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum3 = sum3 + filterData[filterAccessBase + 23] * inTemp3;
		sum4 = sum4 + filterData[filterAccessBase + 18] * inTemp3;
		sum0 = sum0 + filterData[filterAccessBase + 13] * inTemp3;
		sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp3;
		sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum3 = sum3 + filterData[filterAccessBase + 24] * inTemp4;
		sum4 = sum4 + filterData[filterAccessBase + 19] * inTemp4;
		sum0 = sum0 + filterData[filterAccessBase + 14] * inTemp4;
		sum1 = sum1 + filterData[filterAccessBase + 9] * inTemp4;
		sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp4;
		output[outputIdx] = sum3 * alpha + beta;
		outputIdx += outputWidth;

		// 7th row, 12th row
		// convolve with filter 5 times:
		//		1. filter's 5th row (when filter is sliding through the 5th row of input)
		// 		2. filter's 4th row (when filter is sliding through the 6th row of input) 
		//		3. filter's 3rd row (when filter is sliding through the 7th row of input) 
		// 		4. filter's 2nd row (when filter is sliding through the 8th row of input) 
		//		5. filter's 1st row (when filter is sliding through the 9th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum4 = sum4 + filterData[filterAccessBase + 20] * inTemp0;
		sum0 = sum0 + filterData[filterAccessBase + 15] * inTemp0;
		sum1 = sum1 + filterData[filterAccessBase + 10] * inTemp0;
		sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp0;
		sum3 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum4 = sum4 + filterData[filterAccessBase + 21] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 16] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 11] * inTemp1;
		sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp1;
		sum3 = sum3 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum4 = sum4 + filterData[filterAccessBase + 22] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 17] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 12] * inTemp2;
		sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp2;
		sum3 = sum3 + filterData[filterAccessBase + 2] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum4 = sum4 + filterData[filterAccessBase + 23] * inTemp3;
		sum0 = sum0 + filterData[filterAccessBase + 18] * inTemp3;
		sum1 = sum1 + filterData[filterAccessBase + 13] * inTemp3;
		sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp3;
		sum3 = sum3 + filterData[filterAccessBase + 3] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum4 = sum4 + filterData[filterAccessBase + 24] * inTemp4;
		sum0 = sum0 + filterData[filterAccessBase + 19] * inTemp4;
		sum1 = sum1 + filterData[filterAccessBase + 14] * inTemp4;
		sum2 = sum2 + filterData[filterAccessBase + 9] * inTemp4;
		sum3 = sum3 + filterData[filterAccessBase + 4] * inTemp4;
		output[outputIdx] = sum4 * alpha + beta;
		outputIdx += outputWidth;
	}

	// 13th row
	// convolve with filter 4 times:
	//		1. filter's 5th row (when filter is sliding through the 11th row of input)
	//		2. filter's 4th row (when filter is sliding through the 12th row of input)
	// 		3. filter's 3rd row (when filter is sliding through the 13th row of input) 
	//		4. filter's 2nd row (when filter is sliding through the 14th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 20] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 15] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 10] * inTemp0;
	sum3 = sum3 + filterData[filterAccessBase + 5] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 21] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 16] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 11] * inTemp1;
	sum3 = sum3 + filterData[filterAccessBase + 6] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 22] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 17] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 12] * inTemp2;
	sum3 = sum3 + filterData[filterAccessBase + 7] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 23] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 18] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 13] * inTemp3;
	sum3 = sum3 + filterData[filterAccessBase + 8] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 24] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 19] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 14] * inTemp4;
	sum3 = sum3 + filterData[filterAccessBase + 9] * inTemp4;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// 14th row
	// convolve with filter 3 times:
	// 		1. filter's 5th row (when filter is sliding through the 12th row of input) 
	//		2. filter's 4th row (when filter is sliding through the 13th row of input) 
	//		3. filter's 3rd row (when filter is sliding through the 14th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 20] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 15] * inTemp0;
	sum3 = sum3 + filterData[filterAccessBase + 10] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 21] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 16] * inTemp1;
	sum3 = sum3 + filterData[filterAccessBase + 11] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 22] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 17] * inTemp2;
	sum3 = sum3 + filterData[filterAccessBase + 12] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 23] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 18] * inTemp3;
	sum3 = sum3 + filterData[filterAccessBase + 13] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 24] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 19] * inTemp4;
	sum3 = sum3 + filterData[filterAccessBase + 14] * inTemp4;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	output[outputIdx] = sum2 * alpha + beta;
	outputIdx += outputWidth;

	output[outputIdx] = sum3 * alpha + beta;
}

/*
Depthwise Convolution Kernel.

Case: filter 5 x 5, input 14 x 14, stride 2, padding 2

The number of channel must be multiple of 32.
Used in the MobileNet V2 and EfficientNet B0, in case of.
	1)	14 x 14 x 672 -> 14 x 14 x 672, stride = 2, filter = 5
*/
__global__ void Filter5x5_Input14x14_Stride2(const float* input, const float* filter, float* output,
	int inputBatchNumber, int inputChannel, int inputHeight, int inputWidth,
	int filterLayerNumber, int filterHeight, int filterWidth,
	int outputBatchNumber, int outputChannel, int outputHeight, int outputWidth,
	int padding, int stride,
	float alpha, float beta) {

	// every 32 channels is a group.
	__shared__ float filterData[32 * 25];	// filter is 5 x 5 = 25
	__shared__ float inputData[32 * 14 * 18]; // original input is 14 x 14, padded to be 18 x 18. ignore up and bottom padding, so 14 x 18

	float inTemp0, inTemp1, inTemp2, inTemp3, inTemp4;
	float sum0, sum1, sum2;  // to accumulate the row sum result. rolling recycle.

	int channelGroupSize = 32;
	int paddedWidth = inputWidth + 2 * padding;

	// load filter
	int filterLoadSrcIdx = blockIdx.y * channelGroupSize * filterWidth * filterHeight + threadIdx.x;
	if (threadIdx.x < 8 * 25) {
		filterData[threadIdx.x] = filter[filterLoadSrcIdx];	// 8 * 25
		filterData[threadIdx.x + 8 * 25] = filter[filterLoadSrcIdx + 8 * 25]; // 16 * 25
		filterData[threadIdx.x + 8 * 25 * 2] = filter[filterLoadSrcIdx + 8 * 25 * 2]; // 24 * 25
		filterData[threadIdx.x + 8 * 25 * 3] = filter[filterLoadSrcIdx + 8 * 25 * 3]; // 32 * 25, filter loaded
	}

	// set left and right padding
	int leftPaddingIdx = threadIdx.x * paddedWidth;
	// Upper half, left side
	inputData[leftPaddingIdx] = 0;
	inputData[leftPaddingIdx + 1] = 0;
	// Upper half, right side
	inputData[leftPaddingIdx + 16] = 0;
	inputData[leftPaddingIdx + 17] = 0;

	// Bottom half, left side
	inputData[leftPaddingIdx + 16 * inputHeight * paddedWidth] = 0;
	inputData[leftPaddingIdx + 16 * inputHeight * paddedWidth + 1] = 0;
	// Bottom half, right side
	inputData[leftPaddingIdx + 16 * inputHeight * paddedWidth + 16] = 0;
	inputData[leftPaddingIdx + 16 * inputHeight * paddedWidth + 17] = 0;
	__syncthreads();


	// load input
	// for all threads in the same block, use blockIdx.x to find correct batch index, use blockIdx.y to find correct input channel.
	int inputLoadIdxBase = blockIdx.x * inputChannel * inputHeight * inputWidth + blockIdx.y * channelGroupSize * inputHeight * inputWidth;
	int inputLoadSrcIdx = inputLoadIdxBase + threadIdx.x;	// each thread find its own load source.
	int inputLoadDstIdx = (threadIdx.x / inputWidth) * 4 + threadIdx.x + 2;	// each thread find its own load destination.

	inputData[inputLoadDstIdx] = input[inputLoadSrcIdx];
	inputData[inputLoadDstIdx + 16 * 18 * 1] = input[inputLoadSrcIdx + 16 * 14 * 1];
	inputData[inputLoadDstIdx + 16 * 18 * 2] = input[inputLoadSrcIdx + 16 * 14 * 2];
	inputData[inputLoadDstIdx + 16 * 18 * 3] = input[inputLoadSrcIdx + 16 * 14 * 3];
	inputData[inputLoadDstIdx + 16 * 18 * 4] = input[inputLoadSrcIdx + 16 * 14 * 4];
	inputData[inputLoadDstIdx + 16 * 18 * 5] = input[inputLoadSrcIdx + 16 * 14 * 5];
	inputData[inputLoadDstIdx + 16 * 18 * 6] = input[inputLoadSrcIdx + 16 * 14 * 6];
	inputData[inputLoadDstIdx + 16 * 18 * 7] = input[inputLoadSrcIdx + 16 * 14 * 7];
	inputData[inputLoadDstIdx + 16 * 18 * 8] = input[inputLoadSrcIdx + 16 * 14 * 8];
	inputData[inputLoadDstIdx + 16 * 18 * 9] = input[inputLoadSrcIdx + 16 * 14 * 9];
	inputData[inputLoadDstIdx + 16 * 18 * 10] = input[inputLoadSrcIdx + 16 * 14 * 10];
	inputData[inputLoadDstIdx + 16 * 18 * 11] = input[inputLoadSrcIdx + 16 * 14 * 11];
	inputData[inputLoadDstIdx + 16 * 18 * 12] = input[inputLoadSrcIdx + 16 * 14 * 12];
	inputData[inputLoadDstIdx + 16 * 18 * 13] = input[inputLoadSrcIdx + 16 * 14 * 13];
	inputData[inputLoadDstIdx + 16 * 18 * 14] = input[inputLoadSrcIdx + 16 * 14 * 14];
	inputData[inputLoadDstIdx + 16 * 18 * 15] = input[inputLoadSrcIdx + 16 * 14 * 15];
	inputData[inputLoadDstIdx + 16 * 18 * 16] = input[inputLoadSrcIdx + 16 * 14 * 16];
	inputData[inputLoadDstIdx + 16 * 18 * 17] = input[inputLoadSrcIdx + 16 * 14 * 17];
	inputData[inputLoadDstIdx + 16 * 18 * 18] = input[inputLoadSrcIdx + 16 * 14 * 18];
	inputData[inputLoadDstIdx + 16 * 18 * 19] = input[inputLoadSrcIdx + 16 * 14 * 19];
	inputData[inputLoadDstIdx + 16 * 18 * 20] = input[inputLoadSrcIdx + 16 * 14 * 20];
	inputData[inputLoadDstIdx + 16 * 18 * 21] = input[inputLoadSrcIdx + 16 * 14 * 21];
	inputData[inputLoadDstIdx + 16 * 18 * 22] = input[inputLoadSrcIdx + 16 * 14 * 22];
	inputData[inputLoadDstIdx + 16 * 18 * 23] = input[inputLoadSrcIdx + 16 * 14 * 23];
	inputData[inputLoadDstIdx + 16 * 18 * 24] = input[inputLoadSrcIdx + 16 * 14 * 24];
	inputData[inputLoadDstIdx + 16 * 18 * 25] = input[inputLoadSrcIdx + 16 * 14 * 25];
	inputData[inputLoadDstIdx + 16 * 18 * 26] = input[inputLoadSrcIdx + 16 * 14 * 26];
	inputData[inputLoadDstIdx + 16 * 18 * 27] = input[inputLoadSrcIdx + 16 * 14 * 27];
	__syncthreads();

	// convolution
	int outputIdx = blockIdx.x * outputChannel * outputHeight * outputWidth +
		blockIdx.y * channelGroupSize * outputHeight * outputWidth +
		(threadIdx.x / outputWidth) * outputHeight * outputWidth +
		threadIdx.x % outputWidth;

	int inputAccessBase = (threadIdx.x / outputWidth) * paddedWidth * inputHeight + threadIdx.x % outputWidth * 2;
	int filterAccessBase = (threadIdx.x / outputWidth) * filterHeight * filterWidth;
	int inputAccessOffset = 0;

	// 1st row
	// convolve with filter 2 times:
	// 		1. filter's 3rd row (when filter is sliding through the 1st row of input)
	//		2. filter's 1st row (when filter is sliding through the 3rd row of input)
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = filterData[filterAccessBase + 10] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 11] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 12] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 13] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 14] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp4;

	// 2nd row
	// convolve with filter 2 times:
	//		1. filter's 4th row (when filter is sliding through the 1st row of input)
	//		2. filter's 2nd row (when filter is sliding through the 3rd row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 15] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 16] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 17] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 18] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 19] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 9] * inTemp4;

	// 3rd row
	// convolve with filter 3 times:
	//		1. filter's 5th row (when filter is sliding through the 1st row of input)
	//		2. filter's 3rd row (when filter is sliding through the 3rd row of input) 
	//		3. filter's 1st row (when filter is sliding through the 5th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 20] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 10] * inTemp0;
	sum2 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 21] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 11] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 22] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 12] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 23] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 13] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 24] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 14] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp4;
	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// 4th row
	// convolve with filter 2 times:
	// 		1. filter's 4th row (when filter is sliding through the 3rd row of input) 
	//		2. filter's 2nd row (when filter is sliding through the 5th row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 15] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 16] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 17] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 18] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 19] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 9] * inTemp4;

	// 5th row
	// convolve with filter 3 times:
	//		1. filter's 5th row (when filter is sliding through the 3rd row of input)
	//		2. filter's 3rd row (when filter is sliding through the 5th row of input) 
	//		3. filter's 1st row (when filter is sliding through the 7th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 20] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 10] * inTemp0;
	sum0 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 21] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 11] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 22] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 12] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 23] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 13] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 24] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 14] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp4;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// 6th row
	// convolve with filter 2 times:
	// 		1. filter's 4th row (when filter is sliding through the 5th row of input) 
	// 		2. filter's 2nd row (when filter is sliding through the 7th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 15] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 16] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 17] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 18] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 19] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 9] * inTemp4;

	// 7th row
	// convolve with filter 3 times:
	//		1. filter's 5th row (when filter is sliding through the 5th row of input)
	//		2. filter's 3rd row (when filter is sliding through the 7th row of input)
	//		3. filter's 1st row (when filter is sliding through the 9th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 20] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 10] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 21] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 11] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 22] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 12] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 23] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 13] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 24] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 14] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp4;
	output[outputIdx] = sum2 * alpha + beta;
	outputIdx += outputWidth;

	// 8th row
	// convolve with filter 2 times:
	//		1. filter's 4th row (when filter is sliding through the 7th row of input)
	//		2. filter's 2nd row (when filter is sliding through the 9th row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 15] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 16] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 17] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 18] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 19] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 9] * inTemp4;

	// 9th row
	// convolve with filter 3 times:
	//		1. filter's 5th row (when filter is sliding through the 7th row of input)
	//		2. filter's 3rd row (when filter is sliding through the 9th row of input) 
	//		3. filter's 1st row (when filter is sliding through the 11th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 20] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 10] * inTemp0;
	sum2 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 21] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 11] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 22] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 12] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 23] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 13] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 24] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 14] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp4;
	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// 10th row
	// convolve with filter 2 times:
	// 		1. filter's 4th row (when filter is sliding through the 9th row of input) 
	//		2. filter's 2nd row (when filter is sliding through the 11th row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 15] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 16] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 17] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 18] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 19] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 9] * inTemp4;

	// 11th row
	// convolve with filter 3 times:
	//		1. filter's 5th row (when filter is sliding through the 9th row of input)
	//		2. filter's 3rd row (when filter is sliding through the 11th row of input) 
	//		3. filter's 1st row (when filter is sliding through the 13th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 20] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 10] * inTemp0;
	sum0 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 21] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 11] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 22] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 12] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 23] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 13] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 24] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 14] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp4;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// 12th row
	// convolve with filter 2 times:
	// 		1. filter's 4th row (when filter is sliding through the 11th row of input) 
	// 		2. filter's 2nd row (when filter is sliding through the 13th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 15] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 16] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 17] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 18] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 19] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 9] * inTemp4;

	// 13th row
	// convolve with filter 2 times:
	//		1. filter's 5th row (when filter is sliding through the 11th row of input)
	// 		2. filter's 3rd row (when filter is sliding through the 13th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 20] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 10] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 21] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 11] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 22] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 12] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 23] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 13] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 24] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 14] * inTemp4;

	output[outputIdx] = sum2 * alpha + beta;
	outputIdx += outputWidth;

	// 14th row
	// convolve with filter 1 time:
	//		1. filter's 4th row (when filter is sliding through the 13th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 15] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 16] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 17] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 18] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 19] * inTemp4;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;
}

/*
Depthwise Convolution Kernel.

Case: filter 3 x 3, input 28 x 28, stride 1, padding 1

The number of channel must be multiple of 8.
Used in the MobileNet V2 and EfficientNet B0, in case of
	1)	28 x 28 x 240 -> 28 x 28 x 240, stride = 1, filter = 3
*/
__global__ void Filter3x3_Input28x28_Stride1(const float* input, const float* filter, float* output,
	int inputBatchNumber, int inputChannel, int inputHeight, int inputWidth,
	int filterLayerNumber, int filterHeight, int filterWidth,
	int outputBatchNumber, int outputChannel, int outputHeight, int outputWidth,
	int padding, int stride,
	float alpha, float beta) {

	// every 8 channels is a group.
	__shared__ float filterData[8 * 9];	// filter is 3 x 3 = 9
	__shared__ float inputData[8 * 28 * 30]; // original input is 28 x 28, padded to be 30 x 30. ignore up and bottom padding, so 28 x 30

	float inTemp0, inTemp1, inTemp2;
	float sum0, sum1, sum2;  // to accumulate the row sum result. rolling recycle.

	int channelGroupSize = 8;
	int paddedWidth = inputWidth + 2 * padding;

	// load filter
	int filterLoadSrcIdx = blockIdx.y * channelGroupSize * filterWidth * filterHeight + threadIdx.x;
	if (threadIdx.x < 8 * 9) {
		filterData[threadIdx.x] = filter[filterLoadSrcIdx];
	}

	// set left and right padding
	int leftPaddingIdx = threadIdx.x * paddedWidth;
	inputData[leftPaddingIdx] = 0;
	inputData[leftPaddingIdx + 29] = 0; // right side padding

	__syncthreads();

	// load input
	// for all threads in the same block, use blockIdx.x to find correct batch index, use blockIdx.y to find correct input channel.
	int inputLoadIdxBase = blockIdx.x * inputChannel * inputHeight * inputWidth + blockIdx.y * channelGroupSize * inputHeight * inputWidth;
	int inputLoadSrcIdx = inputLoadIdxBase + threadIdx.x;	// each thread find its own load source.
	int inputLoadDstIdx = (threadIdx.x / inputWidth) * 2 + threadIdx.x + 1;	// each thread find its own load destination.

	inputData[inputLoadDstIdx] = input[inputLoadSrcIdx];
	inputData[inputLoadDstIdx + 8 * 30 * 1] = input[inputLoadSrcIdx + 8 * 28 * 1];
	inputData[inputLoadDstIdx + 8 * 30 * 2] = input[inputLoadSrcIdx + 8 * 28 * 2];
	inputData[inputLoadDstIdx + 8 * 30 * 3] = input[inputLoadSrcIdx + 8 * 28 * 3];
	inputData[inputLoadDstIdx + 8 * 30 * 4] = input[inputLoadSrcIdx + 8 * 28 * 4];
	inputData[inputLoadDstIdx + 8 * 30 * 5] = input[inputLoadSrcIdx + 8 * 28 * 5];
	inputData[inputLoadDstIdx + 8 * 30 * 6] = input[inputLoadSrcIdx + 8 * 28 * 6];
	inputData[inputLoadDstIdx + 8 * 30 * 7] = input[inputLoadSrcIdx + 8 * 28 * 7];
	inputData[inputLoadDstIdx + 8 * 30 * 8] = input[inputLoadSrcIdx + 8 * 28 * 8];
	inputData[inputLoadDstIdx + 8 * 30 * 9] = input[inputLoadSrcIdx + 8 * 28 * 9];
	inputData[inputLoadDstIdx + 8 * 30 * 10] = input[inputLoadSrcIdx + 8 * 28 * 10];
	inputData[inputLoadDstIdx + 8 * 30 * 11] = input[inputLoadSrcIdx + 8 * 28 * 11];
	inputData[inputLoadDstIdx + 8 * 30 * 12] = input[inputLoadSrcIdx + 8 * 28 * 12];
	inputData[inputLoadDstIdx + 8 * 30 * 13] = input[inputLoadSrcIdx + 8 * 28 * 13];
	inputData[inputLoadDstIdx + 8 * 30 * 14] = input[inputLoadSrcIdx + 8 * 28 * 14];
	inputData[inputLoadDstIdx + 8 * 30 * 15] = input[inputLoadSrcIdx + 8 * 28 * 15];
	inputData[inputLoadDstIdx + 8 * 30 * 16] = input[inputLoadSrcIdx + 8 * 28 * 16];
	inputData[inputLoadDstIdx + 8 * 30 * 17] = input[inputLoadSrcIdx + 8 * 28 * 17];
	inputData[inputLoadDstIdx + 8 * 30 * 18] = input[inputLoadSrcIdx + 8 * 28 * 18];
	inputData[inputLoadDstIdx + 8 * 30 * 19] = input[inputLoadSrcIdx + 8 * 28 * 19];
	inputData[inputLoadDstIdx + 8 * 30 * 20] = input[inputLoadSrcIdx + 8 * 28 * 20];
	inputData[inputLoadDstIdx + 8 * 30 * 21] = input[inputLoadSrcIdx + 8 * 28 * 21];
	inputData[inputLoadDstIdx + 8 * 30 * 22] = input[inputLoadSrcIdx + 8 * 28 * 22];
	inputData[inputLoadDstIdx + 8 * 30 * 23] = input[inputLoadSrcIdx + 8 * 28 * 23];
	inputData[inputLoadDstIdx + 8 * 30 * 24] = input[inputLoadSrcIdx + 8 * 28 * 24];
	inputData[inputLoadDstIdx + 8 * 30 * 25] = input[inputLoadSrcIdx + 8 * 28 * 25];
	inputData[inputLoadDstIdx + 8 * 30 * 26] = input[inputLoadSrcIdx + 8 * 28 * 26];
	inputData[inputLoadDstIdx + 8 * 30 * 27] = input[inputLoadSrcIdx + 8 * 28 * 27];
	__syncthreads();

	// convolution
	int outputIdx = blockIdx.x * outputChannel * outputHeight * outputWidth +
		blockIdx.y * channelGroupSize * outputHeight * outputWidth +
		(threadIdx.x / outputWidth) * outputHeight * outputWidth +
		threadIdx.x % outputWidth;

	int inputAccessBase = (threadIdx.x / outputWidth) * paddedWidth * inputHeight + threadIdx.x % outputWidth;
	int filterAccessBase = (threadIdx.x / inputWidth) * filterHeight * filterWidth;
	int inputAccessOffset = 0;

	// 1st row
	// convolve with filter 2 times:
	// 		1. filter's 2nd row (when filter is sliding through the 1st row of input) 
	//		2. filter's 1st row (when filter is sliding through the 2nd row of input) 
	inTemp0 = inputData[inputAccessBase];
	sum0 = filterData[filterAccessBase + 3] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1];
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2];
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

#pragma unroll
	for (int i = 0; i < 8; i++) {
		// 2nd row, 5th row, 8th row, 11th row, 14th row, 17th row, 20th row, 23rd row
		// convolve with filter 3 times:
		//		1. filter's 3rd row (when filter is sliding through the 1st row of input)
		// 		2. filter's 2nd row (when filter is sliding through the 2nd row of input) 
		//		3. filter's 1st row (when filter is sliding through the 3rd row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
		sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;
		sum2 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;
		sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;
		sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

		output[outputIdx] = sum0 * alpha + beta;
		outputIdx += outputWidth;

		// 3rd row
		// convolve with filter 3 times:
		//		1. filter's 3rd row (when filter is sliding through the 2nd row of input)
		// 		2. filter's 2nd row (when filter is sliding through the 3rd row of input) 
		//		3. filter's 1st row (when filter is sliding through the 4th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;
		sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp0;
		sum0 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;
		sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;
		sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

		output[outputIdx] = sum1 * alpha + beta;
		outputIdx += outputWidth;

		// 4th row, 7th row
		// convolve with filter three times:
		//		1. filter's 3rd row (when filter is sliding through the 3rd row of input)
		// 		2. filter's 2nd row (when filter is sliding through the 4th row of input) 
		//		3. filter's 1st row (when filter is sliding through the 5th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp0;
		sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp0;
		sum1 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

		output[outputIdx] = sum2 * alpha + beta;
		outputIdx += outputWidth;
	}

	// 26th row
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;
	sum2 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// 27th row
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp0;
	sum0 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// 28th row
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;

	output[outputIdx] = sum2 * alpha + beta;
	outputIdx += outputWidth;

	output[outputIdx] = sum0 * alpha + beta;
}

/*
Depthwise Convolution Kernel.

Case: filter 3 x 3, input 28 x 28, stride 2, padding 1

The number of channel must be multiple of 8.
Used in the MobileNet V2 and EfficientNet B0, in case of
	1)	28 x 28 x 192 -> 14 x 14 x 192, stride = 2, filter = 3
	1)	28 x 28 x 240 -> 14 x 14 x 240, stride = 2, filter = 3
*/
__global__ void Filter3x3_Input28x28_Stride2(const float* input, const float* filter, float* output,
	int inputBatchNumber, int inputChannel, int inputHeight, int inputWidth,
	int filterLayerNumber, int filterHeight, int filterWidth,
	int outputBatchNumber, int outputChannel, int outputHeight, int outputWidth,
	int padding, int stride,
	float alpha, float beta) {

	// every 8 channels is a group.
	__shared__ float filterData[8 * 9];	// filter is 3 x 3 = 9
	__shared__ float inputData[8 * 28 * 30]; // original input is 28 x 28, padded to be 30 x 30. ignore up and bottom padding, so 28 x 30

	float inTemp0, inTemp1, inTemp2;
	float sum0, sum1;  // to accumulate the row sum result. rolling recycle.

	int channelGroupSize = 8;
	int paddedWidth = inputWidth + 2 * padding;

	// load filter
	int filterLoadSrcIdx = blockIdx.y * channelGroupSize * filterWidth * filterHeight + threadIdx.x;
	if (threadIdx.x < 8 * 9) {
		filterData[threadIdx.x] = filter[filterLoadSrcIdx];
	}

	// set left and right padding
	int leftPaddingIdx = threadIdx.x * paddedWidth;
	inputData[leftPaddingIdx] = 0;
	inputData[leftPaddingIdx + 29] = 0;
	inputData[leftPaddingIdx + (channelGroupSize / 2) * inputHeight * paddedWidth] = 0;
	inputData[leftPaddingIdx + (channelGroupSize / 2) * inputHeight * paddedWidth + 29] = 0;

	__syncthreads();

	// load input
	// for all threads in the same block, use blockIdx.x to find correct batch index, use blockIdx.y to find correct input channel.
	int inputLoadIdxBase = blockIdx.x * inputChannel * inputHeight * inputWidth + blockIdx.y * channelGroupSize * inputHeight * inputWidth;
	int inputLoadSrcIdx = inputLoadIdxBase + threadIdx.x;	// each thread find its own load source.
	int inputLoadDstIdx = (threadIdx.x / inputWidth) * 2 + threadIdx.x + 1;	// each thread find its own load destination.

#pragma unroll
	for (int i = 0; i < 56; i++) {
		inputData[inputLoadDstIdx + 4 * 30 * i] = input[inputLoadSrcIdx + 4 * 28 * i];
	}
	__syncthreads();

	// convolution
	int outputIdx = blockIdx.x * outputChannel * outputHeight * outputWidth +
		blockIdx.y * channelGroupSize * outputHeight * outputWidth +
		(threadIdx.x / outputWidth) * outputHeight * outputWidth +
		threadIdx.x % outputWidth;

	int inputAccessBase = (threadIdx.x / outputWidth) * paddedWidth * inputHeight + threadIdx.x % outputWidth * 2;
	int filterAccessBase = (threadIdx.x / outputWidth) * filterHeight * filterWidth;
	int inputAccessOffset = 0;

	// 1st row
	// convolve with filter 1 time:
	// 		1. filter's 2nd row (when filter is sliding through the 1st row of input) 
	inTemp0 = inputData[inputAccessBase];
	sum0 = filterData[filterAccessBase + 3] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1];
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2];
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;

	// 2nd row
	// convolve with filter 2 times:
	//		1. filter's 3rd row (when filter is sliding through the 1st row of input)
	//		2. filter's 1st row (when filter is sliding through the 3rd row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

#pragma unroll
	for (int i = 0; i < 6; i++) {
		// 3rd row
		// convolve with filter 1 time:
		// 		1. filter's 2nd row (when filter is sliding through the 3rd row of input)
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;

		inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;

		inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;

		// 4th row
		// convolve with filter 2 times:
		//		1. filter's 3rd row (when filter is sliding through the 3rd row of input)
		//		2. filter's 1st row (when filter is sliding through the 5th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;
		sum0 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

		output[outputIdx] = sum1 * alpha + beta;
		outputIdx += outputWidth;

		// 5th row
		// convolve with filter 1 time:
		// 		1. filter's 2nd row (when filter is sliding through the 5th row of input)
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp0;

		inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;

		inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;

		// 6th row
		// convolve with filter 2 times:
		//		1. filter's 3rd row (when filter is sliding through the 5th row of input)
		//		2. filter's 1st row (when filter is sliding through the 7th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
		sum1 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

		output[outputIdx] = sum0 * alpha + beta;
		outputIdx += outputWidth;
	}

	// 27th row
	// convolve with filter 1 time:
	// 		1. filter's 2nd row (when filter is sliding through the 27th row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;

	// 28th row
	// convolve with filter 1 time:
	// 		1. filter's 3rd row (when filter is sliding through the 27th row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;
}

/*
Depthwise Convolution Kernel.

Case: filter 5 x 5, input 28 x 28, stride 1, padding 2

The number of channel must be multiple of 8.
Used in the MobileNet V2 and EfficientNet B0, in case of
	1)	28 x 28 x 240 -> 28 x 28 x 240, stride = 1, filter = 5
*/
__global__ void Filter5x5_Input28x28_Stride1(const float* input, const float* filter, float* output,
	int inputBatchNumber, int inputChannel, int inputHeight, int inputWidth,
	int filterLayerNumber, int filterHeight, int filterWidth,
	int outputBatchNumber, int outputChannel, int outputHeight, int outputWidth,
	int padding, int stride,
	float alpha, float beta) {

	// every 8 channels is a group.
	__shared__ float filterData[8 * 25];	// filter is 5 x 5 = 25
	__shared__ float inputData[8 * 28 * 32]; // original input is 28 x 28, padded to be 32 x 32. ignore up and bottom padding, so 28 x 32

	float inTemp0, inTemp1, inTemp2, inTemp3, inTemp4;
	float sum0, sum1, sum2, sum3, sum4;  // to accumulate the row sum result. rolling recycle.
	// cuuint64_t exchange;

	int channelGroupSize = 8;
	// int blockSize = blockDim.x * blockDim.y;
	int paddedWidth = inputWidth + 2 * padding;

	// load filter
	int filterLoadSrcIdx = blockIdx.y * channelGroupSize * filterWidth * filterHeight + threadIdx.x;
	if (threadIdx.x < 8 * 25) {
		filterData[threadIdx.x] = filter[filterLoadSrcIdx];
	}

	// set left and right padding
	int leftPaddingIdx = threadIdx.x * paddedWidth;
	inputData[leftPaddingIdx] = 0;
	inputData[leftPaddingIdx + 1] = 0;
	inputData[leftPaddingIdx + 30] = 0; // right side padding
	inputData[leftPaddingIdx + 31] = 0; // right side padding

	__syncthreads();


	// load input
	// for all threads in the same block, use blockIdx.x to find correct batch index, use blockIdx.y to find correct input channel.
	int inputLoadIdxBase = blockIdx.x * inputChannel * inputHeight * inputWidth + blockIdx.y * channelGroupSize * inputHeight * inputWidth;
	int inputLoadSrcIdx = inputLoadIdxBase + threadIdx.x;	// each thread find its own load source.
	int inputLoadDstIdx = (threadIdx.x / inputWidth) * 4 + threadIdx.x + 2;	// each thread find its own load destination.

	inputData[inputLoadDstIdx] = input[inputLoadSrcIdx];
	inputData[inputLoadDstIdx + 8 * 32 * 1] = input[inputLoadSrcIdx + 32 * 7 * 1];
	inputData[inputLoadDstIdx + 8 * 32 * 2] = input[inputLoadSrcIdx + 32 * 7 * 2];
	inputData[inputLoadDstIdx + 8 * 32 * 3] = input[inputLoadSrcIdx + 32 * 7 * 3];
	inputData[inputLoadDstIdx + 8 * 32 * 4] = input[inputLoadSrcIdx + 32 * 7 * 4];
	inputData[inputLoadDstIdx + 8 * 32 * 5] = input[inputLoadSrcIdx + 32 * 7 * 5];
	inputData[inputLoadDstIdx + 8 * 32 * 6] = input[inputLoadSrcIdx + 32 * 7 * 6];
	inputData[inputLoadDstIdx + 8 * 32 * 7] = input[inputLoadSrcIdx + 32 * 7 * 7];
	inputData[inputLoadDstIdx + 8 * 32 * 8] = input[inputLoadSrcIdx + 32 * 7 * 8];
	inputData[inputLoadDstIdx + 8 * 32 * 9] = input[inputLoadSrcIdx + 32 * 7 * 9];
	inputData[inputLoadDstIdx + 8 * 32 * 10] = input[inputLoadSrcIdx + 32 * 7 * 10];
	inputData[inputLoadDstIdx + 8 * 32 * 11] = input[inputLoadSrcIdx + 32 * 7 * 11];
	inputData[inputLoadDstIdx + 8 * 32 * 12] = input[inputLoadSrcIdx + 32 * 7 * 12];
	inputData[inputLoadDstIdx + 8 * 32 * 13] = input[inputLoadSrcIdx + 32 * 7 * 13];
	inputData[inputLoadDstIdx + 8 * 32 * 14] = input[inputLoadSrcIdx + 32 * 7 * 14];
	inputData[inputLoadDstIdx + 8 * 32 * 15] = input[inputLoadSrcIdx + 32 * 7 * 15];
	inputData[inputLoadDstIdx + 8 * 32 * 16] = input[inputLoadSrcIdx + 32 * 7 * 16];
	inputData[inputLoadDstIdx + 8 * 32 * 17] = input[inputLoadSrcIdx + 32 * 7 * 17];
	inputData[inputLoadDstIdx + 8 * 32 * 18] = input[inputLoadSrcIdx + 32 * 7 * 18];
	inputData[inputLoadDstIdx + 8 * 32 * 19] = input[inputLoadSrcIdx + 32 * 7 * 19];
	inputData[inputLoadDstIdx + 8 * 32 * 20] = input[inputLoadSrcIdx + 32 * 7 * 20];
	inputData[inputLoadDstIdx + 8 * 32 * 21] = input[inputLoadSrcIdx + 32 * 7 * 21];
	inputData[inputLoadDstIdx + 8 * 32 * 22] = input[inputLoadSrcIdx + 32 * 7 * 22];
	inputData[inputLoadDstIdx + 8 * 32 * 23] = input[inputLoadSrcIdx + 32 * 7 * 23];
	inputData[inputLoadDstIdx + 8 * 32 * 24] = input[inputLoadSrcIdx + 32 * 7 * 24];
	inputData[inputLoadDstIdx + 8 * 32 * 25] = input[inputLoadSrcIdx + 32 * 7 * 25];
	inputData[inputLoadDstIdx + 8 * 32 * 26] = input[inputLoadSrcIdx + 32 * 7 * 26];
	inputData[inputLoadDstIdx + 8 * 32 * 27] = input[inputLoadSrcIdx + 32 * 7 * 27];
	__syncthreads();

	// convolution
	int outputIdx = inputLoadIdxBase + (threadIdx.x / inputWidth) * inputHeight * inputWidth + threadIdx.x % inputWidth;

	int inputAccessBase = (threadIdx.x / inputWidth) * paddedWidth * inputHeight + threadIdx.x % inputWidth;
	int filterAccessBase = (threadIdx.x / inputWidth) * filterHeight * filterWidth;
	int inputAccessOffset = 0;
	// 1st row
	// convolve with filter 3 times:
	// 		1. filter's 3rd row (when filter is sliding through the 1st row of input) 
	//		2. filter's 2nd row (when filter is sliding through the 2nd row of input) 
	//		3. filter's 1st row (when filter is sliding through the 3rd row of input)
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = filterData[filterAccessBase + 10] * inTemp0;
	sum1 = filterData[filterAccessBase + 5] * inTemp0;
	sum2 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 11] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 12] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 13] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 14] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 9] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp4;

	// 2nd row
	// convolve with filter 4 times:
	//		1. filter's 4th row (when filter is sliding through the 1st row of input)
	// 		2. filter's 3rd row (when filter is sliding through the 2nd row of input) 
	//		3. filter's 2nd row (when filter is sliding through the 3rd row of input) 
	//		3. filter's 1st row (when filter is sliding through the 4th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 15] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 10] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp0;
	sum3 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 16] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 11] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp1;
	sum3 = sum3 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 17] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 12] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp2;
	sum3 = sum3 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 18] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 13] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp3;
	sum3 = sum3 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 19] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 14] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 9] * inTemp4;
	sum3 = sum3 + filterData[filterAccessBase + 4] * inTemp4;

#pragma unroll
	for (int i = 0; i < 4; i++) {
		// 3rd row
		// convolve with filter 5 times:
		//		1. filter's 5th row (when filter is sliding through the 1st row of input)
		// 		2. filter's 4th row (when filter is sliding through the 2nd row of input) 
		//		3. filter's 3rd row (when filter is sliding through the 3rd row of input) 
		//		4. filter's 2nd row (when filter is sliding through the 4th row of input) 
		//		5. filter's 1st row (when filter is sliding through the 5th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 20] * inTemp0;
		sum1 = sum1 + filterData[filterAccessBase + 15] * inTemp0;
		sum2 = sum2 + filterData[filterAccessBase + 10] * inTemp0;
		sum3 = sum3 + filterData[filterAccessBase + 5] * inTemp0;
		sum4 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 21] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 16] * inTemp1;
		sum2 = sum2 + filterData[filterAccessBase + 11] * inTemp1;
		sum3 = sum3 + filterData[filterAccessBase + 6] * inTemp1;
		sum4 = sum4 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 22] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 17] * inTemp2;
		sum2 = sum2 + filterData[filterAccessBase + 12] * inTemp2;
		sum3 = sum3 + filterData[filterAccessBase + 7] * inTemp2;
		sum4 = sum4 + filterData[filterAccessBase + 2] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 23] * inTemp3;
		sum1 = sum1 + filterData[filterAccessBase + 18] * inTemp3;
		sum2 = sum2 + filterData[filterAccessBase + 13] * inTemp3;
		sum3 = sum3 + filterData[filterAccessBase + 8] * inTemp3;
		sum4 = sum4 + filterData[filterAccessBase + 3] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 24] * inTemp4;
		sum1 = sum1 + filterData[filterAccessBase + 19] * inTemp4;
		sum2 = sum2 + filterData[filterAccessBase + 14] * inTemp4;
		sum3 = sum3 + filterData[filterAccessBase + 9] * inTemp4;
		sum4 = sum4 + filterData[filterAccessBase + 4] * inTemp4;
		output[outputIdx] = sum0 * alpha + beta;
		outputIdx += inputWidth;

		// 4th row
		// convolve with filter 5 times:
		//		1. filter's 5th row (when filter is sliding through the 2nd row of input)
		// 		2. filter's 4th row (when filter is sliding through the 3rd row of input) 
		//		3. filter's 3rd row (when filter is sliding through the 4th row of input) 
		//		4. filter's 2nd row (when filter is sliding through the 5th row of input) 
		//		5. filter's 1st row (when filter is sliding through the 6th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 20] * inTemp0;
		sum2 = sum2 + filterData[filterAccessBase + 15] * inTemp0;
		sum3 = sum3 + filterData[filterAccessBase + 10] * inTemp0;
		sum4 = sum4 + filterData[filterAccessBase + 5] * inTemp0;
		sum0 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 21] * inTemp1;
		sum2 = sum2 + filterData[filterAccessBase + 16] * inTemp1;
		sum3 = sum3 + filterData[filterAccessBase + 11] * inTemp1;
		sum4 = sum4 + filterData[filterAccessBase + 6] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 22] * inTemp2;
		sum2 = sum2 + filterData[filterAccessBase + 17] * inTemp2;
		sum3 = sum3 + filterData[filterAccessBase + 12] * inTemp2;
		sum4 = sum4 + filterData[filterAccessBase + 7] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 23] * inTemp3;
		sum2 = sum2 + filterData[filterAccessBase + 18] * inTemp3;
		sum3 = sum3 + filterData[filterAccessBase + 13] * inTemp3;
		sum4 = sum4 + filterData[filterAccessBase + 8] * inTemp3;
		sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 24] * inTemp4;
		sum2 = sum2 + filterData[filterAccessBase + 19] * inTemp4;
		sum3 = sum3 + filterData[filterAccessBase + 14] * inTemp4;
		sum4 = sum4 + filterData[filterAccessBase + 9] * inTemp4;
		sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp4;
		output[outputIdx] = sum1 * alpha + beta;
		outputIdx += inputWidth;

		// 5th row
		// convolve with filter 5 times:
		//		1. filter's 5th row (when filter is sliding through the 3rd row of input)
		// 		2. filter's 4th row (when filter is sliding through the 4th row of input) 
		//		3. filter's 3rd row (when filter is sliding through the 5th row of input) 
		// 		4. filter's 2nd row (when filter is sliding through the 6th row of input) 
		//		5. filter's 1st row (when filter is sliding through the 7th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 20] * inTemp0;
		sum3 = sum3 + filterData[filterAccessBase + 15] * inTemp0;
		sum4 = sum4 + filterData[filterAccessBase + 10] * inTemp0;
		sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp0;
		sum1 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 21] * inTemp1;
		sum3 = sum3 + filterData[filterAccessBase + 16] * inTemp1;
		sum4 = sum4 + filterData[filterAccessBase + 11] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 22] * inTemp2;
		sum3 = sum3 + filterData[filterAccessBase + 17] * inTemp2;
		sum4 = sum4 + filterData[filterAccessBase + 12] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 23] * inTemp3;
		sum3 = sum3 + filterData[filterAccessBase + 18] * inTemp3;
		sum4 = sum4 + filterData[filterAccessBase + 13] * inTemp3;
		sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp3;
		sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 24] * inTemp4;
		sum3 = sum3 + filterData[filterAccessBase + 19] * inTemp4;
		sum4 = sum4 + filterData[filterAccessBase + 14] * inTemp4;
		sum0 = sum0 + filterData[filterAccessBase + 9] * inTemp4;
		sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp4;

		output[outputIdx] = sum2 * alpha + beta;
		outputIdx += inputWidth;

		// 6th row
		// convolve with filter 5 times:
		//		1. filter's 5th row (when filter is sliding through the 4th row of input)
		// 		2. filter's 4th row (when filter is sliding through the 5th row of input) 
		//		3. filter's 3rd row (when filter is sliding through the 6th row of input) 
		// 		4. filter's 2nd row (when filter is sliding through the 7th row of input) 
		//		5. filter's 1st row (when filter is sliding through the 8th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum3 = sum3 + filterData[filterAccessBase + 20] * inTemp0;
		sum4 = sum4 + filterData[filterAccessBase + 15] * inTemp0;
		sum0 = sum0 + filterData[filterAccessBase + 10] * inTemp0;
		sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp0;
		sum2 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum3 = sum3 + filterData[filterAccessBase + 21] * inTemp1;
		sum4 = sum4 + filterData[filterAccessBase + 16] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 11] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp1;
		sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum3 = sum3 + filterData[filterAccessBase + 22] * inTemp2;
		sum4 = sum4 + filterData[filterAccessBase + 17] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 12] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp2;
		sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum3 = sum3 + filterData[filterAccessBase + 23] * inTemp3;
		sum4 = sum4 + filterData[filterAccessBase + 18] * inTemp3;
		sum0 = sum0 + filterData[filterAccessBase + 13] * inTemp3;
		sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp3;
		sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum3 = sum3 + filterData[filterAccessBase + 24] * inTemp4;
		sum4 = sum4 + filterData[filterAccessBase + 19] * inTemp4;
		sum0 = sum0 + filterData[filterAccessBase + 14] * inTemp4;
		sum1 = sum1 + filterData[filterAccessBase + 9] * inTemp4;
		sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp4;
		output[outputIdx] = sum3 * alpha + beta;
		outputIdx += inputWidth;

		// 7th row
		// convolve with filter 5 times:
		//		1. filter's 5th row (when filter is sliding through the 5th row of input)
		// 		2. filter's 4th row (when filter is sliding through the 6th row of input) 
		//		3. filter's 3rd row (when filter is sliding through the 7th row of input) 
		// 		4. filter's 2nd row (when filter is sliding through the 8th row of input) 
		//		5. filter's 1st row (when filter is sliding through the 9th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum4 = sum4 + filterData[filterAccessBase + 20] * inTemp0;
		sum0 = sum0 + filterData[filterAccessBase + 15] * inTemp0;
		sum1 = sum1 + filterData[filterAccessBase + 10] * inTemp0;
		sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp0;
		sum3 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum4 = sum4 + filterData[filterAccessBase + 21] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 16] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 11] * inTemp1;
		sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp1;
		sum3 = sum3 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum4 = sum4 + filterData[filterAccessBase + 22] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 17] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 12] * inTemp2;
		sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp2;
		sum3 = sum3 + filterData[filterAccessBase + 2] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum4 = sum4 + filterData[filterAccessBase + 23] * inTemp3;
		sum0 = sum0 + filterData[filterAccessBase + 18] * inTemp3;
		sum1 = sum1 + filterData[filterAccessBase + 13] * inTemp3;
		sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp3;
		sum3 = sum3 + filterData[filterAccessBase + 3] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum4 = sum4 + filterData[filterAccessBase + 24] * inTemp4;
		sum0 = sum0 + filterData[filterAccessBase + 19] * inTemp4;
		sum1 = sum1 + filterData[filterAccessBase + 14] * inTemp4;
		sum2 = sum2 + filterData[filterAccessBase + 9] * inTemp4;
		sum3 = sum3 + filterData[filterAccessBase + 4] * inTemp4;
		output[outputIdx] = sum4 * alpha + beta;
		outputIdx += inputWidth;
	}

	// 23rd row
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 20] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 15] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 10] * inTemp0;
	sum3 = sum3 + filterData[filterAccessBase + 5] * inTemp0;
	sum4 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 21] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 16] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 11] * inTemp1;
	sum3 = sum3 + filterData[filterAccessBase + 6] * inTemp1;
	sum4 = sum4 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 22] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 17] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 12] * inTemp2;
	sum3 = sum3 + filterData[filterAccessBase + 7] * inTemp2;
	sum4 = sum4 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 23] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 18] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 13] * inTemp3;
	sum3 = sum3 + filterData[filterAccessBase + 8] * inTemp3;
	sum4 = sum4 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 24] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 19] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 14] * inTemp4;
	sum3 = sum3 + filterData[filterAccessBase + 9] * inTemp4;
	sum4 = sum4 + filterData[filterAccessBase + 4] * inTemp4;
	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += inputWidth;

	// 24th row
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 20] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 15] * inTemp0;
	sum3 = sum3 + filterData[filterAccessBase + 10] * inTemp0;
	sum4 = sum4 + filterData[filterAccessBase + 5] * inTemp0;
	sum0 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 21] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 16] * inTemp1;
	sum3 = sum3 + filterData[filterAccessBase + 11] * inTemp1;
	sum4 = sum4 + filterData[filterAccessBase + 6] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 22] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 17] * inTemp2;
	sum3 = sum3 + filterData[filterAccessBase + 12] * inTemp2;
	sum4 = sum4 + filterData[filterAccessBase + 7] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 23] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 18] * inTemp3;
	sum3 = sum3 + filterData[filterAccessBase + 13] * inTemp3;
	sum4 = sum4 + filterData[filterAccessBase + 8] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 24] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 19] * inTemp4;
	sum3 = sum3 + filterData[filterAccessBase + 14] * inTemp4;
	sum4 = sum4 + filterData[filterAccessBase + 9] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp4;
	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += inputWidth;

	// 25th row
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 20] * inTemp0;
	sum3 = sum3 + filterData[filterAccessBase + 15] * inTemp0;
	sum4 = sum4 + filterData[filterAccessBase + 10] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 21] * inTemp1;
	sum3 = sum3 + filterData[filterAccessBase + 16] * inTemp1;
	sum4 = sum4 + filterData[filterAccessBase + 11] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 22] * inTemp2;
	sum3 = sum3 + filterData[filterAccessBase + 17] * inTemp2;
	sum4 = sum4 + filterData[filterAccessBase + 12] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 23] * inTemp3;
	sum3 = sum3 + filterData[filterAccessBase + 18] * inTemp3;
	sum4 = sum4 + filterData[filterAccessBase + 13] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 24] * inTemp4;
	sum3 = sum3 + filterData[filterAccessBase + 19] * inTemp4;
	sum4 = sum4 + filterData[filterAccessBase + 14] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 9] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp4;

	output[outputIdx] = sum2 * alpha + beta;
	outputIdx += inputWidth;

	// 26th row
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum3 = sum3 + filterData[filterAccessBase + 20] * inTemp0;
	sum4 = sum4 + filterData[filterAccessBase + 15] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 10] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp0;
	sum2 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum3 = sum3 + filterData[filterAccessBase + 21] * inTemp1;
	sum4 = sum4 + filterData[filterAccessBase + 16] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 11] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum3 = sum3 + filterData[filterAccessBase + 22] * inTemp2;
	sum4 = sum4 + filterData[filterAccessBase + 17] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 12] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum3 = sum3 + filterData[filterAccessBase + 23] * inTemp3;
	sum4 = sum4 + filterData[filterAccessBase + 18] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 13] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum3 = sum3 + filterData[filterAccessBase + 24] * inTemp4;
	sum4 = sum4 + filterData[filterAccessBase + 19] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 14] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 9] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp4;
	output[outputIdx] = sum3 * alpha + beta;
	outputIdx += inputWidth;

	// 27th row
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum4 = sum4 + filterData[filterAccessBase + 20] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 15] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 10] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum4 = sum4 + filterData[filterAccessBase + 21] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 16] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 11] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum4 = sum4 + filterData[filterAccessBase + 22] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 17] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 12] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum4 = sum4 + filterData[filterAccessBase + 23] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 18] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 13] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum4 = sum4 + filterData[filterAccessBase + 24] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 19] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 14] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 9] * inTemp4;
	output[outputIdx] = sum4 * alpha + beta;
	outputIdx += inputWidth;

	// 28th row
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 20] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 15] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 10] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 21] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 16] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 11] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 22] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 17] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 12] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 23] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 18] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 13] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 24] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 19] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 14] * inTemp4;
	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += inputWidth;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += inputWidth;

	output[outputIdx] = sum2 * alpha + beta;
	outputIdx += inputWidth;
}

/*
Depthwise Convolution Kernel.

Case: filter 3 x 3, input 56 x 56, stride 1, padding 1

Used in the MobileNet V2 and EfficientNet B0, in case of
	1)	56 x 56 x 144 -> 56 x 56 x 144, stride = 1, filter = 3
*/
__global__ void Filter3x3_Input56x56_Stride1(const float* input, const float* filter, float* output,
	int inputBatchNumber, int inputChannel, int inputHeight, int inputWidth,
	int filterLayerNumber, int filterHeight, int filterWidth,
	int outputBatchNumber, int outputChannel, int outputHeight, int outputWidth,
	int padding, int stride,
	float alpha, float beta) {

	__shared__ float filterData[9];	// filter is 3 x 3 = 9
	__shared__ float inputData[58 * 58]; // original input is 56 x 56, padded to be 58 x 58.

	float inTemp0, inTemp1, inTemp2;
	float sum0, sum1, sum2;  // to accumulate the row sum result. rolling recycle.

	int channelGroupSize = 1;
	int paddedWidth = inputWidth + 2 * padding;

	// load filter
	int filterLoadSrcIdx = blockIdx.y * channelGroupSize * filterWidth * filterHeight + threadIdx.x;
	if (threadIdx.x < 9) {
		filterData[threadIdx.x] = filter[filterLoadSrcIdx];
	}

	// set padding
	if (threadIdx.x >= 32 && threadIdx.x < 88) {
		int leftPaddingIdx = (threadIdx.x - 31) * 58;
		inputData[leftPaddingIdx] = 0;
		inputData[leftPaddingIdx + 57] = 0;
	}
	if (threadIdx.x >= 96 && threadIdx.x < 154) {
		inputData[threadIdx.x - 96] = 0;
	}
	if (threadIdx.x >= 160 && threadIdx.x < 218) {
		inputData[threadIdx.x - 160 + 58 * 57] = 0;
	}
	__syncthreads();

	// load input
	// for all threads in the same block, use blockIdx.x to find correct batch index, use blockIdx.y to find correct input channel.
	int inputLoadIdxBase = blockIdx.x * inputChannel * inputHeight * inputWidth + blockIdx.y * channelGroupSize * inputHeight * inputWidth;
	int inputLoadSrcIdx = inputLoadIdxBase + threadIdx.x;	// each thread find its own load source.
	int inputLoadDstIdx = (threadIdx.x / inputWidth) * 2 + threadIdx.x + 1 + paddedWidth;	// each thread find its own load destination.

#pragma unroll
	for (int i = 0; i < 14; i++) {
		inputData[inputLoadDstIdx + 4 * 58 * i] = input[inputLoadSrcIdx + 4 * 56 * i];
	}

	__syncthreads();

	// convolution
	int outputIdx = inputLoadIdxBase + (threadIdx.x / inputWidth) * 14 * inputWidth + threadIdx.x % inputWidth;

	// 4 * 56 threads are separated to 4 groups.
	// first group handles 1 - 14 row
	// second group handles 15 - 28 row
	// third group handles 29 - 42 row
	// forth group handles 43 - 56 row
	int inputAccessBase = (threadIdx.x / inputWidth) * paddedWidth * 14 + threadIdx.x % inputWidth;
	// int filterAccessBase = (threadIdx.x / inputWidth) * filterHeight * filterWidth;
	int filterAccessBase = 0;
	int inputAccessOffset = 0;

	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = filterData[filterAccessBase] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;


	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;


#pragma unroll
	for (int i = 0; i < 4; i++) {
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
		sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;
		sum2 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;
		sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;
		sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

		output[outputIdx] = sum0 * alpha + beta;
		outputIdx += inputWidth;

		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;
		sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp0;
		sum0 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;
		sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;
		sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

		output[outputIdx] = sum1 * alpha + beta;
		outputIdx += inputWidth;

		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp0;
		sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp0;
		sum1 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

		output[outputIdx] = sum2 * alpha + beta;
		outputIdx += inputWidth;
	}
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += inputWidth;

	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;

	output[outputIdx] = sum1 * alpha + beta;
}

/*
Depthwise Convolution Kernel.

Case: filter 3 x 3, input 56 x 56, stride 2, padding 1

Used in the MobileNet V2 and EfficientNet B0, in case of
	1)	56 x 56 x 144 -> 28 x 28 x 144, stride = 2, filter = 3
*/
__global__ void Filter3x3_Input56x56_Stride2(const float* input, const float* filter, float* output,
	int inputBatchNumber, int inputChannel, int inputHeight, int inputWidth,
	int filterLayerNumber, int filterHeight, int filterWidth,
	int outputBatchNumber, int outputChannel, int outputHeight, int outputWidth,
	int padding, int stride,
	float alpha, float beta) {

	__shared__ float filterData[2 * 9];	// filter is 3 x 3 = 9
	__shared__ float inputData[2 * 56 * 58]; // original input is 56 x 56, padded to be 58 x 58. ignore up and bottom padding, so 56 x 58

	float inTemp0, inTemp1, inTemp2;
	float sum0, sum1;  // to accumulate the row sum result. rolling recycle.

	int channelGroupSize = 2;
	int paddedWidth = inputWidth + 2 * padding;

	// load filter
	int filterLoadSrcIdx = blockIdx.y * channelGroupSize * filterWidth * filterHeight + threadIdx.x;
	if (threadIdx.x < 2 * 9)
	{
		filterData[threadIdx.x] = filter[filterLoadSrcIdx];
	}

	// set padding
	int leftPaddingIdx = threadIdx.x * paddedWidth;
	inputData[leftPaddingIdx] = 0;
	inputData[leftPaddingIdx + 57] = 0;
	inputData[leftPaddingIdx + (channelGroupSize / 2) * inputHeight * paddedWidth] = 0;
	inputData[leftPaddingIdx + (channelGroupSize / 2) * inputHeight * paddedWidth + 57] = 0;

	__syncthreads();

	// load input
	// for all threads in the same block, use blockIdx.x to find correct batch index, use blockIdx.y to find correct input channel.
	int inputLoadIdxBase = blockIdx.x * inputChannel * inputHeight * inputWidth + blockIdx.y * channelGroupSize * inputHeight * inputWidth;
	int inputLoadSrcIdx = inputLoadIdxBase + threadIdx.x;	// each thread find its own load source.
	int inputLoadDstIdx = (threadIdx.x / inputWidth) * 2 + threadIdx.x + 1;	// each thread find its own load destination.

#pragma unroll
	for (int i = 0; i < 112; i++) {
		inputData[inputLoadDstIdx + 58 * i] = input[inputLoadSrcIdx + 56 * i];
	}

	__syncthreads();

	// convolution
	int outputIdx = blockIdx.x * outputChannel * outputHeight * outputWidth +
		blockIdx.y * channelGroupSize * outputHeight * outputWidth +
		(threadIdx.x / outputWidth) * outputHeight * outputWidth + threadIdx.x % outputWidth;

	int inputAccessBase = (threadIdx.x / outputWidth) * paddedWidth * inputHeight + threadIdx.x % outputWidth * 2;
	int filterAccessBase = (threadIdx.x / outputWidth) * filterHeight * filterWidth;
	int inputAccessOffset = 0;

	// 1st row
	// convolve with filter 1 time:
	// 		1. filter's 2nd row (when filter is sliding through the 1st row of input) 
	inTemp0 = inputData[inputAccessBase];
	sum0 = filterData[filterAccessBase + 3] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1];
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2];
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;

	// 2nd row
	// convolve with filter 2 times:
	//		1. filter's 3rd row (when filter is sliding through the 1st row of input)
	//		2. filter's 1st row (when filter is sliding through the 3rd row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

#pragma unroll
	for (int i = 0; i < 13; i++) {
		// 3rd row
		// convolve with filter 1 time:
		// 		1. filter's 2nd row (when filter is sliding through the 3rd row of input)
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;

		inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;

		inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;

		// 4th row
		// convolve with filter 2 times:
		//		1. filter's 3rd row (when filter is sliding through the 3rd row of input)
		//		2. filter's 1st row (when filter is sliding through the 5th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;
		sum0 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

		output[outputIdx] = sum1 * alpha + beta;
		outputIdx += outputWidth;

		// 5th row
		// convolve with filter 1 time:
		// 		1. filter's 2nd row (when filter is sliding through the 5th row of input)
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp0;

		inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp1;

		inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp2;

		// 6th row
		// convolve with filter 2 times:
		//		1. filter's 3rd row (when filter is sliding through the 5th row of input)
		//		2. filter's 1st row (when filter is sliding through the 7th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp0;
		sum1 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

		output[outputIdx] = sum0 * alpha + beta;
		outputIdx += outputWidth;
	}

	// 55th row
	// convolve with filter 1 time:
	// 		1. filter's 2nd row (when filter is sliding through the 27th row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp2;

	// 56th row
	// convolve with filter 1 time:
	// 		1. filter's 3rd row (when filter is sliding through the 27th row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp0;

	inTemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp1;

	inTemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp2;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;
}

/*
Depthwise Convolution Kernel.

Case: filter 5 x 5, input 56 x 56, stride 2, padding 2

Used in the MobileNet V2 and EfficientNet B0, in case of
	1)	56 x 56 x 144 -> 28 x 28 x 144, stride = 2, filter = 5
*/
__global__ void Filter5x5_Input56x56_Stride2(const float* input, const float* filter, float* output,
	int inputBatchNumber, int inputChannel, int inputHeight, int inputWidth,
	int filterLayerNumber, int filterHeight, int filterWidth,
	int outputBatchNumber, int outputChannel, int outputHeight, int outputWidth,
	int padding, int stride,
	float alpha, float beta) {

	__shared__ float filterData[2 * 25];	// filter is 5 x 5 = 25
	__shared__ float inputData[2 * 56 * 60]; // original input is 56 x 56, padded to be 60 x 60. ignore up and bottom padding, so 56 x 60

	float inTemp0, inTemp1, inTemp2, inTemp3, inTemp4;
	float sum0, sum1, sum2;  // to accumulate the row sum result. rolling recycle.

	int channelGroupSize = 2;
	int paddedWidth = inputWidth + 2 * padding;

	// load filter
	int filterLoadSrcIdx = blockIdx.y * channelGroupSize * filterWidth * filterHeight + threadIdx.x;
	if (threadIdx.x < 2 * 25) {
		filterData[threadIdx.x] = filter[filterLoadSrcIdx];
	}

	// set padding
	int leftPaddingIdx = threadIdx.x * paddedWidth;
	inputData[leftPaddingIdx] = 0;
	inputData[leftPaddingIdx + 1] = 0;

	inputData[leftPaddingIdx + paddedWidth - 2] = 0;
	inputData[leftPaddingIdx + paddedWidth - 1] = 0;

	inputData[leftPaddingIdx + (channelGroupSize / 2) * inputHeight * paddedWidth] = 0;
	inputData[leftPaddingIdx + (channelGroupSize / 2) * inputHeight * paddedWidth + 1] = 0;

	inputData[leftPaddingIdx + (channelGroupSize / 2) * inputHeight * paddedWidth + paddedWidth - 2] = 0;
	inputData[leftPaddingIdx + (channelGroupSize / 2) * inputHeight * paddedWidth + paddedWidth - 1] = 0;

	__syncthreads();

	// load input
	// for all threads in the same block, use blockIdx.x to find correct batch index, use blockIdx.y to find correct input channel.
	int inputLoadIdxBase = blockIdx.x * inputChannel * inputHeight * inputWidth + blockIdx.y * channelGroupSize * inputHeight * inputWidth;
	int inputLoadSrcIdx = inputLoadIdxBase + threadIdx.x;	// each thread find its own load source.
	int inputLoadDstIdx = (threadIdx.x / inputWidth) * 4 + threadIdx.x + 2;	// each thread find its own load destination.

#pragma unroll
	for (int i = 0; i < 112; i++) {
		inputData[inputLoadDstIdx + 60 * i] = input[inputLoadSrcIdx + 56 * i];
	}

	__syncthreads();

	// convolution
	int outputIdx = blockIdx.x * outputChannel * outputHeight * outputWidth +
		blockIdx.y * channelGroupSize * outputHeight * outputWidth +
		(threadIdx.x / outputWidth) * outputHeight * outputWidth + threadIdx.x % outputWidth;

	int inputAccessBase = (threadIdx.x / outputWidth) * paddedWidth * inputHeight + threadIdx.x % outputWidth * 2;
	int filterAccessBase = (threadIdx.x / outputWidth) * filterHeight * filterWidth;
	int inputAccessOffset = 0;

	// 1st row
	// convolve with filter 2 times:
	// 		1. filter's 3rd row (when filter is sliding through the 1st row of input)
	//		2. filter's 1st row (when filter is sliding through the 3rd row of input)
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = filterData[filterAccessBase + 10] * inTemp0;
	sum1 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 11] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 12] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 13] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 14] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp4;

	// 2nd row
	// convolve with filter 2 times:
	//		1. filter's 4th row (when filter is sliding through the 1st row of input)
	//		2. filter's 2nd row (when filter is sliding through the 3rd row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 15] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 16] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 17] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 18] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 19] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 9] * inTemp4;

#pragma unroll
	for (int i = 0; i < 8; i++) {
		// 3rd row, 45
		// convolve with filter 3 times:
		//		1. filter's 5th row (when filter is sliding through the 1st row of input)
		//		2. filter's 3rd row (when filter is sliding through the 3rd row of input) 
		//		3. filter's 1st row (when filter is sliding through the 5th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 20] * inTemp0;
		sum1 = sum1 + filterData[filterAccessBase + 10] * inTemp0;
		sum2 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 21] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 11] * inTemp1;
		sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 22] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 12] * inTemp2;
		sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 23] * inTemp3;
		sum1 = sum1 + filterData[filterAccessBase + 13] * inTemp3;
		sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 24] * inTemp4;
		sum1 = sum1 + filterData[filterAccessBase + 14] * inTemp4;
		sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp4;
		output[outputIdx] = sum0 * alpha + beta;
		outputIdx += outputWidth;

		// 4th row
		// convolve with filter 2 times:
		// 		1. filter's 4th row (when filter is sliding through the 3rd row of input) 
		//		2. filter's 2nd row (when filter is sliding through the 5th row of input)
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 15] * inTemp0;
		sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 16] * inTemp1;
		sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 17] * inTemp2;
		sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 18] * inTemp3;
		sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 19] * inTemp4;
		sum2 = sum2 + filterData[filterAccessBase + 9] * inTemp4;

		// 5th row
		// convolve with filter 3 times:
		//		1. filter's 5th row (when filter is sliding through the 3rd row of input)
		//		2. filter's 3rd row (when filter is sliding through the 5th row of input) 
		//		3. filter's 1st row (when filter is sliding through the 7th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 20] * inTemp0;
		sum2 = sum2 + filterData[filterAccessBase + 10] * inTemp0;
		sum0 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 21] * inTemp1;
		sum2 = sum2 + filterData[filterAccessBase + 11] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 22] * inTemp2;
		sum2 = sum2 + filterData[filterAccessBase + 12] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 23] * inTemp3;
		sum2 = sum2 + filterData[filterAccessBase + 13] * inTemp3;
		sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum1 = sum1 + filterData[filterAccessBase + 24] * inTemp4;
		sum2 = sum2 + filterData[filterAccessBase + 14] * inTemp4;
		sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp4;

		output[outputIdx] = sum1 * alpha + beta;
		outputIdx += outputWidth;

		// 6th row
		// convolve with filter 2 times:
		// 		1. filter's 4th row (when filter is sliding through the 5th row of input) 
		// 		2. filter's 2nd row (when filter is sliding through the 7th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 15] * inTemp0;
		sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 16] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 17] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 18] * inTemp3;
		sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 19] * inTemp4;
		sum0 = sum0 + filterData[filterAccessBase + 9] * inTemp4;

		// 7th row
		// convolve with filter 3 times:
		//		1. filter's 5th row (when filter is sliding through the 5th row of input)
		//		2. filter's 3rd row (when filter is sliding through the 7th row of input)
		//		3. filter's 1st row (when filter is sliding through the 9th row of input) 
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 20] * inTemp0;
		sum0 = sum0 + filterData[filterAccessBase + 10] * inTemp0;
		sum1 = filterData[filterAccessBase + 0] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 21] * inTemp1;
		sum0 = sum0 + filterData[filterAccessBase + 11] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 1] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 22] * inTemp2;
		sum0 = sum0 + filterData[filterAccessBase + 12] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 2] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 23] * inTemp3;
		sum0 = sum0 + filterData[filterAccessBase + 13] * inTemp3;
		sum1 = sum1 + filterData[filterAccessBase + 3] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum2 = sum2 + filterData[filterAccessBase + 24] * inTemp4;
		sum0 = sum0 + filterData[filterAccessBase + 14] * inTemp4;
		sum1 = sum1 + filterData[filterAccessBase + 4] * inTemp4;
		output[outputIdx] = sum2 * alpha + beta;
		outputIdx += outputWidth;

		// 8th row, 50th row
		// convolve with filter 2 times:
		//		1. filter's 4th row (when filter is sliding through the 7th row of input)
		//		2. filter's 2nd row (when filter is sliding through the 9th row of input)
		inputAccessOffset += paddedWidth;
		inTemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 15] * inTemp0;
		sum1 = sum1 + filterData[filterAccessBase + 5] * inTemp0;

		inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 16] * inTemp1;
		sum1 = sum1 + filterData[filterAccessBase + 6] * inTemp1;

		inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 17] * inTemp2;
		sum1 = sum1 + filterData[filterAccessBase + 7] * inTemp2;

		inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 18] * inTemp3;
		sum1 = sum1 + filterData[filterAccessBase + 8] * inTemp3;

		inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
		sum0 = sum0 + filterData[filterAccessBase + 19] * inTemp4;
		sum1 = sum1 + filterData[filterAccessBase + 9] * inTemp4;
	}


	// 51st row
	// convolve with filter 3 times:
	//		1. filter's 5th row (when filter is sliding through the 49th row of input)
	//		2. filter's 3rd row (when filter is sliding through the 51th row of input) 
	//		3. filter's 1st row (when filter is sliding through the 53rd row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 20] * inTemp0;
	sum1 = sum1 + filterData[filterAccessBase + 10] * inTemp0;
	sum2 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 21] * inTemp1;
	sum1 = sum1 + filterData[filterAccessBase + 11] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 22] * inTemp2;
	sum1 = sum1 + filterData[filterAccessBase + 12] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 23] * inTemp3;
	sum1 = sum1 + filterData[filterAccessBase + 13] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 24] * inTemp4;
	sum1 = sum1 + filterData[filterAccessBase + 14] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 4] * inTemp4;
	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// 52nd row
	// convolve with filter 2 times:
	// 		1. filter's 4th row (when filter is sliding through the 51st row of input) 
	//		2. filter's 2nd row (when filter is sliding through the 53rd row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 15] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 5] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 16] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 6] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 17] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 7] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 18] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 8] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 19] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 9] * inTemp4;

	// 53rd row
	// convolve with filter 3 times:
	//		1. filter's 5th row (when filter is sliding through the 51st row of input)
	//		2. filter's 3rd row (when filter is sliding through the 53rd row of input) 
	//		3. filter's 1st row (when filter is sliding through the 55th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 20] * inTemp0;
	sum2 = sum2 + filterData[filterAccessBase + 10] * inTemp0;
	sum0 = filterData[filterAccessBase + 0] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 21] * inTemp1;
	sum2 = sum2 + filterData[filterAccessBase + 11] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 1] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 22] * inTemp2;
	sum2 = sum2 + filterData[filterAccessBase + 12] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 2] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 23] * inTemp3;
	sum2 = sum2 + filterData[filterAccessBase + 13] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 3] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum1 = sum1 + filterData[filterAccessBase + 24] * inTemp4;
	sum2 = sum2 + filterData[filterAccessBase + 14] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 4] * inTemp4;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// 54th row
	// convolve with filter 2 times:
	// 		1. filter's 4th row (when filter is sliding through the 53rd row of input) 
	// 		2. filter's 2nd row (when filter is sliding through the 55th row of input) 
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 15] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 5] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 16] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 6] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 17] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 7] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 18] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 8] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 19] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 9] * inTemp4;

	// 55th row
	// convolve with filter 2 times:
	//		1. filter's 5th row (when filter is sliding through the 53th row of input)
	//		2. filter's 3rd row (when filter is sliding through the 55th row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 20] * inTemp0;
	sum0 = sum0 + filterData[filterAccessBase + 10] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 21] * inTemp1;
	sum0 = sum0 + filterData[filterAccessBase + 11] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 22] * inTemp2;
	sum0 = sum0 + filterData[filterAccessBase + 12] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 23] * inTemp3;
	sum0 = sum0 + filterData[filterAccessBase + 13] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum2 = sum2 + filterData[filterAccessBase + 24] * inTemp4;
	sum0 = sum0 + filterData[filterAccessBase + 14] * inTemp4;
	output[outputIdx] = sum2 * alpha + beta;
	outputIdx += outputWidth;

	// 56th row
	// convolve with filter 1 times:
	//		1. filter's 4th row (when filter is sliding through the 55th row of input)
	inputAccessOffset += paddedWidth;
	inTemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 15] * inTemp0;

	inTemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 16] * inTemp1;

	inTemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 17] * inTemp2;

	inTemp3 = inputData[inputAccessBase + 3 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 18] * inTemp3;

	inTemp4 = inputData[inputAccessBase + 4 + inputAccessOffset];
	sum0 = sum0 + filterData[filterAccessBase + 19] * inTemp4;
	output[outputIdx] = sum0 * alpha + beta;
}

/*
Depthwise Convolution Kernel.

Case: filter 3 x 3, input 112 x 112, stride 1, padding 1


Used in the MobileNet V2 and EfficientNet B0, in case of
	1)	112 x 112 x 32 -> 112 x 112 x 32, stride = 1, filter = 3
*/
__global__ void Filter3x3_Input112x112_Stride1(const float* input, const float* filter, float* output,
	int inputBatchNumber, int inputChannel, int inputHeight, int inputWidth,
	int filterLayerNumber, int filterHeight, int filterWidth,
	int outputBatchNumber, int outputChannel, int outputHeight, int outputWidth,
	int padding, int stride,
	float alpha, float beta) {

	// filter is 3 x 3. 9 elements in total
	__shared__ float filterData[9];
	// 4 blocks handle one 112 x 112 input. Each block handles 28 rows. With padding, each row has 114 elements.
	__shared__ float inputData[31 * 114];

	float intemp0, intemp1, intemp2;
	float sum0, sum1, sum2;

	int paddedWidth = inputWidth + 2 * padding;
	int blockGroup = 4;

	// load filter
	int filterLoadSrcIdx = blockIdx.y / blockGroup * filterHeight * filterWidth + threadIdx.x;
	if (threadIdx.x < filterWidth * filterHeight) {
		filterData[threadIdx.x] = filter[filterLoadSrcIdx];
	}

	int leftPaddingIdx = 0;
	// set padding
	if (threadIdx.x >= 32 && threadIdx.x < 62) {
		leftPaddingIdx = (threadIdx.x - 32) * paddedWidth;
		inputData[leftPaddingIdx] = 0;						// left padding
		inputData[leftPaddingIdx + paddedWidth - 1] = 0;	// right padding
	}
	if (threadIdx.x >= 112) {
		inputData[threadIdx.x - 111] = 0;					// Top padding
		inputData[threadIdx.x - 111 + 29 * paddedWidth] = 0;// Bottom padding
	}
	__syncthreads();

	int inputLoadIdxBase = blockIdx.x * inputHeight * inputWidth * inputChannel +
		blockIdx.y / blockGroup * inputWidth * inputHeight +
		(blockIdx.y & 3) * inputHeight / blockGroup * inputWidth;

	// block 0 needs to process 28 rows + bottom 1 row, no upper padding.
	// block 1 needs to process 28 rows + upper 1 row + bottom 1 row
	// block 2 needs to process 28 rows + upper 1 row + bottom 1 row
	// block 3 needs to process 28 rows + upper 1 row, no bottom padding
	int inputLoadSrcIdx = inputLoadIdxBase + threadIdx.x - inputWidth;
	int inputLoadDstIdx = (threadIdx.x / inputWidth) * 2 + threadIdx.x + 1;
	if ((blockIdx.y & 3) == 0) {
		inputLoadSrcIdx += inputWidth;
		inputLoadDstIdx += paddedWidth;
	}

	// each block load 28 rows, and each time load 2 rows, so 14 times
#pragma unroll
	for (int i = 0; i < 14; i++) {
		inputData[inputLoadDstIdx + 2 * 114 * i] = input[inputLoadSrcIdx + 2 * 112 * i];
	}
	// block3 do not need to load extra 1 bottom row. 
	if ((blockIdx.y & 3) != 3) {
		inputData[inputLoadDstIdx + 2 * 114 * 14] = input[inputLoadSrcIdx + 2 * 112 * 14];

	} else {
		if (threadIdx.x < 112) {
			inputData[inputLoadDstIdx + 2 * 114 * 14] = input[inputLoadSrcIdx + 2 * 112 * 14];
		}
	}
	__syncthreads();

	// for 224 threads in a block, first 112 threads process first 14 rows, second 112 threads process rest of the 14 rows
	int outputIdx = blockIdx.x * outputHeight * outputWidth * outputChannel +
		(blockIdx.y / blockGroup) * outputHeight * outputWidth +
		(blockIdx.y & 3) * (outputHeight / blockGroup) * outputWidth +
		(threadIdx.x / outputWidth) * 14 * outputWidth + threadIdx.x % outputWidth;

	int inputAccessBase = (threadIdx.x / inputWidth) * 14 * paddedWidth + threadIdx.x % inputWidth;
	int inputAccessOffset = 0;

	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = filterData[0] * intemp0;
	intemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[1] * intemp1;
	intemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[2] * intemp2;

	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[3] * intemp0;
	sum1 = filterData[0] * intemp0;
	intemp1 = inputData[inputAccessBase + 1 + inputAccessOffset];
	sum0 = sum0 + filterData[4] * intemp1;
	sum1 = sum1 + filterData[1] * intemp1;
	intemp2 = inputData[inputAccessBase + 2 + inputAccessOffset];
	sum0 = sum0 + filterData[5] * intemp2;
	sum1 = sum1 + filterData[2] * intemp2;

#pragma unroll
	for (int i = 0; i < 4; i++) {
		inputAccessOffset += paddedWidth;
		intemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum0 = sum0 + filterData[6] * intemp0;
		sum1 = sum1 + filterData[3] * intemp0;
		sum2 = filterData[0] * intemp0;
		intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum0 = sum0 + filterData[7] * intemp1;
		sum1 = sum1 + filterData[4] * intemp1;
		sum2 = sum2 + filterData[1] * intemp1;
		intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum0 = sum0 + filterData[8] * intemp2;
		sum1 = sum1 + filterData[5] * intemp2;
		sum2 = sum2 + filterData[2] * intemp2;

		output[outputIdx] = sum0 * alpha + beta;
		outputIdx += outputWidth;

		inputAccessOffset += paddedWidth;
		intemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum1 = sum1 + filterData[6] * intemp0;
		sum2 = sum2 + filterData[3] * intemp0;
		sum0 = filterData[0] * intemp0;
		intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum1 = sum1 + filterData[7] * intemp1;
		sum2 = sum2 + filterData[4] * intemp1;
		sum0 = sum0 + filterData[1] * intemp1;
		intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum1 = sum1 + filterData[8] * intemp2;
		sum2 = sum2 + filterData[5] * intemp2;
		sum0 = sum0 + filterData[2] * intemp2;

		output[outputIdx] = sum1 * alpha + beta;
		outputIdx += outputWidth;

		inputAccessOffset += paddedWidth;
		intemp0 = inputData[inputAccessBase + inputAccessOffset];
		sum2 = sum2 + filterData[6] * intemp0;
		sum0 = sum0 + filterData[3] * intemp0;
		sum1 = filterData[0] * intemp0;
		intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
		sum2 = sum2 + filterData[7] * intemp1;
		sum0 = sum0 + filterData[4] * intemp1;
		sum1 = sum1 + filterData[1] * intemp1;
		intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
		sum2 = sum2 + filterData[8] * intemp2;
		sum0 = sum0 + filterData[5] * intemp2;
		sum1 = sum1 + filterData[2] * intemp2;

		output[outputIdx] = sum2 * alpha + beta;
		outputIdx += outputWidth;
	}

	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[6] * intemp0;
	sum1 = sum1 + filterData[3] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[7] * intemp1;
	sum1 = sum1 + filterData[4] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[8] * intemp2;
	sum1 = sum1 + filterData[5] * intemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += inputWidth;

	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[6] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[7] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[8] * intemp2;

	output[outputIdx] = sum1 * alpha + beta;
}

/*
Depthwise Convolution Kernel.

Case: filter 3 x 3, input 112 x 112, stride 2, padding 1
2 blocks process 1 channel.

Used in the MobileNet V2 and EfficientNet B0, in case of
	1)	112 x 112 x 96 -> 56 x 56 x 96, stride = 2, filter = 3
*/

__global__ void Filter3x3_Input112x112_Stride2(const float* input, const float* filter, float* output,
	int inputBatchNumber, int inputChannel, int inputHeight, int inputWidth,
	int filterLayerNumber, int filterHeight, int filterWidth,
	int outputBatchNumber, int outputChannel, int outputHeight, int outputWidth,
	int padding, int stride,
	float alpha, float beta) {

	// filter is 3 x 3. 9 elements in total
	__shared__ float filterData[9];
	// 2 blocks handle one 112 x 112 input. Each block handles 56 rows. With padding, 58 rows, each row has 114 elements
	__shared__ float inputData[59 * 114];

	float intemp0, intemp1, intemp2;
	float sum0, sum1;

	int paddedWidth = inputWidth + 2 * padding;
	int blockGroup = 2;

	// load filter
	int filterLoadSrcIdx = blockIdx.y / blockGroup * filterHeight * filterWidth + threadIdx.x;
	if (threadIdx.x < filterWidth * filterHeight) {
		filterData[threadIdx.x] = filter[filterLoadSrcIdx];
	}

	int leftPaddingIdx = 0;
	// set padding
	if (threadIdx.x >= 32 && threadIdx.x < 90) {
		leftPaddingIdx = (threadIdx.x - 32) * paddedWidth;
		inputData[leftPaddingIdx] = 0;						// left padding
		inputData[leftPaddingIdx + paddedWidth - 1] = 0;	// right padding
	}
	if (threadIdx.x >= 112) {
		inputData[threadIdx.x - 111] = 0;					// Top padding
		inputData[threadIdx.x - 111 + 57 * paddedWidth] = 0;// Bottom padding
	}
	__syncthreads();

	int inputLoadIdxBase = blockIdx.x * inputHeight * inputWidth * inputChannel +
		blockIdx.y / blockGroup * inputHeight * inputWidth +
		(blockIdx.y & 1) * inputHeight / blockGroup * inputWidth;

	// block 0 needs to process 56 rows + bottom 1 row, no upper padding.
	// block 1 needs to process 56 rows + upper 1 row + bottom 1 row
	int inputLoadSrcIdx = inputLoadIdxBase + threadIdx.x - inputWidth;
	int inputLoadDstIdx = (threadIdx.x / inputWidth) * 2 + threadIdx.x + 1;
	if ((blockIdx.y & 1) == 0) {
		inputLoadSrcIdx += inputWidth;
		inputLoadDstIdx += paddedWidth;
	}

	// each block load 56 rows, and each time load 2 rows, so 28 times
#pragma unroll
	for (int i = 0; i < 28; i++) {
		inputData[inputLoadDstIdx + 2 * 114 * i] = input[inputLoadSrcIdx + 2 * 112 * i];
	}
	// block1 do not need to load extra 1 bottom row. 
	if ((blockIdx.y & 1) != 1) {
		inputData[inputLoadDstIdx + 2 * 114 * 28] = input[inputLoadSrcIdx + 2 * 112 * 28];
	}
	else {
		if (threadIdx.x < 112) {
			inputData[inputLoadDstIdx + 2 * 114 * 28] = input[inputLoadSrcIdx + 2 * 112 * 28];
		}
	}
	__syncthreads();

	// for a 224-thread block, every 56-thread group processes 14 rows in the input, and write 7 rows in the output
	int outputIdx = blockIdx.x * outputHeight * outputWidth * outputChannel +
		(blockIdx.y / blockGroup) * outputHeight * outputWidth +
		(blockIdx.y & 1) * (outputHeight / blockGroup) * outputWidth +
		(threadIdx.x / outputWidth) * 7 * outputWidth + 
		threadIdx.x % outputWidth;

	int inputAccessBase = (threadIdx.x / outputWidth) * 14 * paddedWidth + threadIdx.x % outputWidth * 2;
	int inputAccessOffset = 0;

	// row 0
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 =	filterData[0] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[1] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[2] * intemp2;

	// row 1
	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[3] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[4] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[5] * intemp2;

	// row 2
	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[6] * intemp0;
	sum1 = filterData[0] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[7] * intemp1;
	sum1 = sum1 + filterData[1] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[8] * intemp2;
	sum1 = sum1 + filterData[2] * intemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// row 3
	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[3] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[4] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[5] * intemp2;

	// row 4
	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = filterData[0] * intemp0;
	sum1 = sum1 + filterData[6] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[1] * intemp1;
	sum1 = sum1 + filterData[7] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[2] * intemp2;
	sum1 = sum1 + filterData[8] * intemp2;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// row 5
	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[3] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[4] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[5] * intemp2;

	// row 6
	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[6] * intemp0;
	sum1 = filterData[0] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[7] * intemp1;
	sum1 = sum1 + filterData[1] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[8] * intemp2;
	sum1 = sum1 + filterData[2] * intemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// row 7
	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[3] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[4] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[5] * intemp2;

	// row 8
	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[6] * intemp0;
	sum0 = filterData[0] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[7] * intemp1;
	sum0 = sum0 + filterData[1] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[8] * intemp2;
	sum0 = sum0 + filterData[2] * intemp2;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// row 9
	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[3] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[4] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[5] * intemp2;

	// row 10
	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[6] * intemp0;
	sum1 = filterData[0] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[7] * intemp1;
	sum1 = sum1 + filterData[1] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[8] * intemp2;
	sum1 = sum1 + filterData[2] * intemp2;

	output[outputIdx] = sum0 * alpha + beta;
	outputIdx += outputWidth;

	// row 11
	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[3] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[4] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[5] * intemp2;

	// row 12
	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum1 = sum1 + filterData[6] * intemp0;
	sum0 = filterData[0] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum1 = sum1 + filterData[7] * intemp1;
	sum0 = sum0 + filterData[1] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum1 = sum1 + filterData[8] * intemp2;
	sum0 = sum0 + filterData[2] * intemp2;

	output[outputIdx] = sum1 * alpha + beta;
	outputIdx += outputWidth;

	// row 13
	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[3] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[4] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[5] * intemp2;

	// row 14
	inputAccessOffset += paddedWidth;
	intemp0 = inputData[inputAccessBase + inputAccessOffset];
	sum0 = sum0 + filterData[6] * intemp0;
	intemp1 = inputData[inputAccessBase + inputAccessOffset + 1];
	sum0 = sum0 + filterData[7] * intemp1;
	intemp2 = inputData[inputAccessBase + inputAccessOffset + 2];
	sum0 = sum0 + filterData[8] * intemp2;

	output[outputIdx] = sum0 * alpha + beta;
}

/*
To test depthwise convolution kernels.
*/
int main(int argc, char* argv[]) {
	// GPU warm up for benchmarking
	warmup << <128, 128 >> > ();

	// Input dimension
	int inputBatchNumber = 0;
	int inputChannel = 0;
	int inputHeight = 0;
	int inputWidth = 0;

	// Filter dimension
	int filterLayerNumber = 0;
	int filterChannel = 0;
	int filterHeight = 0;
	int filterWidth = 0;

	// Output dimension
	int outputBatchNumber = 0;
	int outputChannel = 0;
	int outputHeight = 0;
	int outputWidth = 0;

	// padding on height and width
	int paddingHeight = 0;
	int paddingWidth = 0;

	// stride
	int stride = 1;

	float alpha = 1.0;
	float beta = 0.0;

	// Initialize all required parameters
	// Input dimensions
	inputBatchNumber = atoi(argv[1]);
	inputChannel = atoi(argv[2]);
	inputHeight = atoi(argv[3]);
	inputWidth = inputHeight;           // Assume that inputs are square

	// Filter dimensions
	filterLayerNumber = inputChannel;
	filterChannel = 1;
	filterHeight = atoi(argv[4]);
	filterWidth = filterHeight;         // Assume that filters are square

	// Padding size
	if (filterWidth == 3) {
		paddingHeight = 1;
		paddingWidth = 1;
	}
	else if (filterWidth == 5) {
		paddingHeight = 2;
		paddingWidth = 2;
	}

	// Stride
	stride = atoi(argv[5]);

	// Output dimensions
	outputBatchNumber = inputBatchNumber;
	outputChannel = inputChannel;
	outputHeight = (inputHeight + paddingHeight * 2 - filterHeight) / stride + 1;
	outputWidth = (inputWidth + paddingWidth * 2 - filterWidth) / stride + 1;

	// Data size
	int inputSize = inputBatchNumber * inputChannel * inputHeight * inputWidth;
	int filterSize = filterLayerNumber * filterChannel * filterHeight * filterWidth;
	int outputSize = outputBatchNumber * outputChannel * outputHeight * outputWidth;

	// allocate host memory and device memory for input data, and copy it from host to device.
	float* hostInput = (float*)malloc(inputSize * sizeof(float));
	srand(time(NULL));
	for (int i = 0; i < inputSize; i++) {
		hostInput[i] = (float)(float(rand()) / float((RAND_MAX)) * 5.0);
	}
	float* deviceInput;
	checkCuda(hipMalloc((void**)&deviceInput, inputSize * sizeof(float)));
	checkCuda(hipMemcpy(deviceInput, hostInput, inputSize * sizeof(float), hipMemcpyHostToDevice));

	// allocate host memory and device memory for filter data, and copy it from host to device.
	float* hostFilter = (float*)malloc(filterSize * sizeof(float));
	srand(time(NULL));
	for (int i = 0; i < filterSize; i++) {
		hostFilter[i] = (float)(float(rand()) / float((RAND_MAX)) * 5.0);
	}
	float* deviceFilter;
	checkCuda(hipMalloc((void**)&deviceFilter, filterSize * sizeof(float)));
	checkCuda(hipMemcpy(deviceFilter, hostFilter, filterSize * sizeof(float), hipMemcpyHostToDevice));

	// allocate host memory and device memory for kernel output data
	float* hostKernelOutput = (float*)malloc(outputSize * sizeof(float));
	float* deviceKernelOutput;
	checkCuda(hipMalloc((void**)&deviceKernelOutput, outputSize * sizeof(float)));

	// allocate host memory and device memory for Cudnn output data
	float* hostCudnnOutput = (float*)malloc(outputSize * sizeof(float));
	float* deviceCudnnOutput;
	checkCuda(hipMalloc((void**)&deviceCudnnOutput, outputSize * sizeof(float)));

	// Use Cuda event to measure running time
	float elapsedTime = 0.0;
	float kernelTime = 0.0;
	float cudnnTime = 0.0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Kernel Invocation
	if (stride == 1) {
		if (filterHeight == 3) {
			if (inputHeight == 7) {
				dim3 gridSize(outputBatchNumber, outputChannel / 32);
				dim3 blockSize(7 * 32, 1);
				hipEventRecord(start);
				Filter3x3_Input7x7_Stride1 << <gridSize, blockSize >> > (
					deviceInput, deviceFilter, deviceKernelOutput,
					inputBatchNumber, inputChannel, inputHeight, inputWidth,
					filterLayerNumber, filterHeight, filterWidth,
					outputBatchNumber, outputChannel, outputHeight, outputWidth,
					paddingWidth, stride,
					alpha, beta);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				kernelTime = elapsedTime;
				printf("Elapsed Time for Depthwise Convolution Filter 3x3 Input 7x7 Stride 1: %f ms.\n", elapsedTime);
			}
			else if (inputHeight == 14) {
				dim3 gridSize(outputBatchNumber, outputChannel / 16);
				dim3 blockSize(14 * 16, 1);
				hipEventRecord(start);
				Filter3x3_Input14x14_Stride1 << <gridSize, blockSize >> > (
					deviceInput, deviceFilter, deviceKernelOutput,
					inputBatchNumber, inputChannel, inputHeight, inputWidth,
					filterLayerNumber, filterHeight, filterWidth,
					outputBatchNumber, outputChannel, outputHeight, outputWidth,
					paddingWidth, stride,
					alpha, beta);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				kernelTime = elapsedTime;
				printf("Elapsed Time for Depthwise Convolution Filter 3x3 Input 14x14 Stride 1: %f ms.\n", elapsedTime);
			}
			else if (inputHeight == 28) {
				dim3 gridSize(outputBatchNumber, outputChannel / 8);
				dim3 blockSize(28 * 8, 1);
				hipEventRecord(start);
				Filter3x3_Input28x28_Stride1 << <gridSize, blockSize >> > (
					deviceInput, deviceFilter, deviceKernelOutput,
					inputBatchNumber, inputChannel, inputHeight, inputWidth,
					filterLayerNumber, filterHeight, filterWidth,
					outputBatchNumber, outputChannel, outputHeight, outputWidth,
					paddingWidth, stride,
					alpha, beta);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				kernelTime = elapsedTime;
				printf("Elapsed Time for Depthwise Convolution Filter 3x3 Input 28x28 Stride 1: %f ms.\n", elapsedTime);
			}
			else if (inputHeight == 56) {
				dim3 gridSize(outputBatchNumber, outputChannel);
				dim3 blockSize(4 * 56, 1);
				hipEventRecord(start);
				Filter3x3_Input56x56_Stride1 << <gridSize, blockSize >> > (
					deviceInput, deviceFilter, deviceKernelOutput,
					inputBatchNumber, inputChannel, inputHeight, inputWidth,
					filterLayerNumber, filterHeight, filterWidth,
					outputBatchNumber, outputChannel, outputHeight, outputWidth,
					paddingWidth, stride,
					alpha, beta);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				kernelTime = elapsedTime;
				printf("Elapsed Time for Depthwise Convolution Filter 3x3 Input 56x56 Stride 1: %f ms.\n", elapsedTime);
			}
			else if (inputHeight == 112) {
				dim3 gridSize(outputBatchNumber, outputChannel * 4);
				dim3 blockSize(2 * 112, 1);
				hipEventRecord(start);
				Filter3x3_Input112x112_Stride1 << <gridSize, blockSize >> > (
					deviceInput, deviceFilter, deviceKernelOutput,
					inputBatchNumber, inputChannel, inputHeight, inputWidth,
					filterLayerNumber, filterHeight, filterWidth,
					outputBatchNumber, outputChannel, outputHeight, outputWidth,
					paddingWidth, stride,
					alpha, beta);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				kernelTime = elapsedTime;
				printf("Elapsed Time for Depthwise Convolution Filter 3x3 Input 112x112 Stride 1: %f ms.\n", elapsedTime);
			}
		}
		else if (filterHeight == 5) {
			if (inputHeight == 7) {
				dim3 gridSize(outputBatchNumber, outputChannel / 32);
				dim3 blockSize(7 * 32, 1);
				hipEventRecord(start);
				Filter5x5_Input7x7_Stride1 << <gridSize, blockSize >> > (
					deviceInput, deviceFilter, deviceKernelOutput,
					inputBatchNumber, inputChannel, inputHeight, inputWidth,
					filterLayerNumber, filterHeight, filterWidth,
					outputBatchNumber, outputChannel, outputHeight, outputWidth,
					paddingWidth, stride,
					alpha, beta);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				kernelTime = elapsedTime;
				printf("Elapsed Time for Depthwise Convolution Filter 5x5 Input 7x7 Stride 1: %f ms.\n", elapsedTime);
			}
			else if (inputHeight == 14) {
				dim3 gridSize(outputBatchNumber, outputChannel / 16);
				dim3 blockSize(14 * 16, 1);
				hipEventRecord(start);
				Filter5x5_Input14x14_Stride1 << <gridSize, blockSize >> > (
					deviceInput, deviceFilter, deviceKernelOutput,
					inputBatchNumber, inputChannel, inputHeight, inputWidth,
					filterLayerNumber, filterHeight, filterWidth,
					outputBatchNumber, outputChannel, outputHeight, outputWidth,
					paddingWidth, stride,
					alpha, beta);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				kernelTime = elapsedTime;
				printf("Elapsed Time for Depthwise Convolution Filter 5x5 Input 14x14 Stride 1: %f ms.\n", elapsedTime);
			}
			else if (inputHeight == 28) {
				dim3 gridSize(outputBatchNumber, outputChannel / 8);
				dim3 blockSize(28 * 8, 1);
				hipEventRecord(start);
				Filter5x5_Input28x28_Stride1 << <gridSize, blockSize >> > (
					deviceInput, deviceFilter, deviceKernelOutput,
					inputBatchNumber, inputChannel, inputHeight, inputWidth,
					filterLayerNumber, filterHeight, filterWidth,
					outputBatchNumber, outputChannel, outputHeight, outputWidth,
					paddingWidth, stride,
					alpha, beta);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				kernelTime = elapsedTime;
				printf("Elapsed Time for Depthwise Convolution Filter 5x5 Input 28x28 Stride 1: %f ms.\n", elapsedTime);
			}
		}
	}
	else if (stride == 2) {
		if (filterHeight == 3) {
			if (inputHeight == 14) {
				dim3 gridSize(outputBatchNumber, outputChannel / 32);
				dim3 blockSize(7 * 32, 1);
				hipEventRecord(start);
				Filter3x3_Input14x14_Stride2 << <gridSize, blockSize >> > (
					deviceInput, deviceFilter, deviceKernelOutput,
					inputBatchNumber, inputChannel, inputHeight, inputWidth,
					filterLayerNumber, filterHeight, filterWidth,
					outputBatchNumber, outputChannel, outputHeight, outputWidth,
					paddingWidth, stride,
					alpha, beta);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				kernelTime = elapsedTime;
				printf("Elapsed Time for Depthwise Convolution Filter 3x3 Input 14x14 Stride 2: %f ms.\n", elapsedTime);
			}
			else if (inputHeight == 28) {
				dim3 gridSize(outputBatchNumber, outputChannel / 8); // if channel group size = 16, shared memory exceeded.
				dim3 blockSize(14 * 8, 1);
				hipEventRecord(start);
				Filter3x3_Input28x28_Stride2 << <gridSize, blockSize >> > (
					deviceInput, deviceFilter, deviceKernelOutput,
					inputBatchNumber, inputChannel, inputHeight, inputWidth,
					filterLayerNumber, filterHeight, filterWidth,
					outputBatchNumber, outputChannel, outputHeight, outputWidth,
					paddingWidth, stride,
					alpha, beta);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				kernelTime = elapsedTime;
				printf("Elapsed Time for Depthwise Convolution Filter 3x3 Input 28x28 Stride 2: %f ms.\n", elapsedTime);
			}
			else if (inputHeight == 56) {
				dim3 gridSize(outputBatchNumber, outputChannel / 2);
				dim3 blockSize(28 * 2, 1);
				hipEventRecord(start);
				Filter3x3_Input56x56_Stride2 << <gridSize, blockSize >> > (
					deviceInput, deviceFilter, deviceKernelOutput,
					inputBatchNumber, inputChannel, inputHeight, inputWidth,
					filterLayerNumber, filterHeight, filterWidth,
					outputBatchNumber, outputChannel, outputHeight, outputWidth,
					paddingWidth, stride,
					alpha, beta);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				kernelTime = elapsedTime;
				printf("Elapsed Time for Depthwise Convolution Filter 3x3 Input 56x56 Stride 2: %f ms.\n", elapsedTime);
			}
			else if (inputHeight == 112) {
				dim3 gridSize(outputBatchNumber, outputChannel * 2);
				dim3 blockSize(56 * 4, 1);
				hipEventRecord(start);
				Filter3x3_Input112x112_Stride2 << <gridSize, blockSize >> > (
					deviceInput, deviceFilter, deviceKernelOutput,
					inputBatchNumber, inputChannel, inputHeight, inputWidth,
					filterLayerNumber, filterHeight, filterWidth,
					outputBatchNumber, outputChannel, outputHeight, outputWidth,
					paddingWidth, stride,
					alpha, beta);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				kernelTime = elapsedTime;
				printf("Elapsed Time for Depthwise Convolution Filter 3x3 Input 112x112 Stride 2: %f ms.\n", elapsedTime);
			}
		}
		else if (filterHeight == 5) {
			if (inputHeight == 14) {
				dim3 gridSize(outputBatchNumber, outputChannel / 32);
				dim3 blockSize(7 * 32, 1);
				hipEventRecord(start);
				Filter5x5_Input14x14_Stride2 << <gridSize, blockSize >> > (
					deviceInput, deviceFilter, deviceKernelOutput,
					inputBatchNumber, inputChannel, inputHeight, inputWidth,
					filterLayerNumber, filterHeight, filterWidth,
					outputBatchNumber, outputChannel, outputHeight, outputWidth,
					paddingWidth, stride,
					alpha, beta);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				kernelTime = elapsedTime;
				printf("Elapsed Time for Depthwise Convolution Filter 5x5 Input 14x14 Stride 2: %f ms.\n", elapsedTime);
			}
			else if (inputHeight == 56) {
				dim3 gridSize(outputBatchNumber, outputChannel / 2);
				dim3 blockSize(28 * 2, 1);
				hipEventRecord(start);
				Filter5x5_Input56x56_Stride2 << <gridSize, blockSize >> > (
					deviceInput, deviceFilter, deviceKernelOutput,
					inputBatchNumber, inputChannel, inputHeight, inputWidth,
					filterLayerNumber, filterHeight, filterWidth,
					outputBatchNumber, outputChannel, outputHeight, outputWidth,
					paddingWidth, stride,
					alpha, beta);
				hipEventRecord(stop);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&elapsedTime, start, stop);
				kernelTime = elapsedTime;
				printf("Elapsed Time for Depthwise Convolution Filter 5x5 Input 56x56 Stride 2: %f ms.\n", elapsedTime);
			}
		}
	}

	// Copy kernel output from device to host
	checkCuda(hipMemcpy(hostKernelOutput, deviceKernelOutput, outputSize * sizeof(float), hipMemcpyDeviceToHost));

	// Create cudnn
	hipdnnHandle_t cudnn;
	checkCudnn(hipdnnCreate(&cudnn));

	// input descriptor
	hipdnnTensorDescriptor_t inputDesc;
	checkCudnn(hipdnnCreateTensorDescriptor(&inputDesc));
	checkCudnn(hipdnnSetTensor4dDescriptor(inputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, inputBatchNumber, inputChannel, inputHeight, inputWidth));

	// filter descriptor
	hipdnnFilterDescriptor_t filterDesc;
	checkCudnn(hipdnnCreateFilterDescriptor(&filterDesc));
	checkCudnn(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, filterLayerNumber, filterChannel, filterHeight, filterWidth));

	// output descriptor
	hipdnnTensorDescriptor_t outputDesc;
	checkCudnn(hipdnnCreateTensorDescriptor(&outputDesc));
	checkCudnn(hipdnnSetTensor4dDescriptor(outputDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, outputBatchNumber, outputChannel, outputHeight, outputWidth));

	// convolution descriptor
	hipdnnConvolutionDescriptor_t convDesc;
	checkCudnn(hipdnnCreateConvolutionDescriptor(&convDesc));
	// dilation is 1
	checkCudnn(hipdnnSetConvolution2dDescriptor(convDesc, paddingHeight, paddingWidth, stride, stride, 1, 1, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
	checkCudnn(hipdnnSetConvolutionGroupCount(convDesc, inputChannel));

	// set algorithm
	int returnedAlgoCount = 0;
	hipdnnConvolutionFwdAlgoPerf_t perfResults;
	hipdnnFindConvolutionForwardAlgorithm(cudnn,
		inputDesc,
		filterDesc,
		convDesc,
		outputDesc,
		1,
		&returnedAlgoCount,
		&perfResults);
	hipdnnConvolutionFwdAlgo_t algo = perfResults.algo;
	// hipdnnConvolutionFwdAlgo_t algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;

	// create workspace
	size_t workspaceSize = 0;
	void* workspaceData = nullptr;
	checkCudnn(hipdnnGetConvolutionForwardWorkspaceSize(cudnn, inputDesc, filterDesc, convDesc, outputDesc, algo, &workspaceSize));
	checkCuda(hipMalloc(&workspaceData, workspaceSize));

	// Use CUDNN to check kernel result and measure running time
	hipEventRecord(start);
	hipdnnConvolutionForward(
		cudnn,
		&alpha,
		inputDesc,
		deviceInput,
		filterDesc,
		deviceFilter,
		convDesc,
		algo,
		workspaceData,
		workspaceSize,
		&beta,
		outputDesc,
		deviceCudnnOutput
	);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Elapsed Time for : %f ms.\n", elapsedTime);
	cudnnTime = elapsedTime;
	// Copy Cudnn result from device to host
	checkCuda(hipMemcpy(hostCudnnOutput, deviceCudnnOutput, outputSize * sizeof(float), hipMemcpyDeviceToHost));

	writeCsv(inputBatchNumber, inputChannel, inputHeight, filterHeight, stride, kernelTime, cudnnTime);

	// Compare Kernel result and Cudnn result
	if (compareOutput(outputBatchNumber, outputChannel, outputHeight, outputWidth, hostKernelOutput, hostCudnnOutput, 0.1) == 0) {
		printf("Kernel Calculation Correct.\n");
	}

	free(hostInput);
	free(hostFilter);
	free(hostKernelOutput);
	free(hostCudnnOutput);

	hipFree(deviceInput);
	hipFree(deviceFilter);
	hipFree(deviceKernelOutput);
	hipFree(deviceCudnnOutput);

	hipdnnDestroy(cudnn);
	hipdnnDestroyTensorDescriptor(inputDesc);
	hipdnnDestroyTensorDescriptor(outputDesc);
	hipdnnDestroyConvolutionDescriptor(convDesc);
	hipdnnDestroyFilterDescriptor(filterDesc);
	hipFree(workspaceData);

	checkCuda(hipDeviceReset());
	return 0;
}